
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>
#include <time.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#define TEST_RUN 10 
#define ESP 10e-10
using namespace std;


void check_cuda_error(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}

void check_C(float * dC, int m, int n, float * checkC) {
  for (int i = 0; i < m * n; i++){
    //cout << i << endl;
    if (fabs(dC[i] - checkC[i]) > ESP){
      cout << "error:" << fabs(dC[i] - checkC[i]) << endl;
      return;
    }
  }
  cout << "correct" << endl;
}


float test_cublas_mm(int m, int n, int k, 
            __half * dA, int lda, 
            __half * dB, int ldb, 
            __half * dC, int ldc);


void test(int m, int k);

int main(){
  for (int i = 10240; i <= 30720; i += 1024){
  //int i = 1024;
    cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 2 << ")" << endl;
    test(i, i);
  }
}

void test(int m, int k){
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 2;
    //int k = 20480;
    __half * A = new __half[m * k];
    __half * B = new __half[n * k];
    __half * C = new __half[m * n];
    __half * checkC = new __half[m * n];     

    for (int i = 0; i < m * k; i++){
    	A[i] = __float2half((float)i/(m * k));
    }

    for (int i = 0; i < n * k; i++){
    	B[i] = __float2half((float)i/(n * k));
    }
    
    __half * dA;
    hipMalloc(&dA, m * k * sizeof(__half));
    int lda = m;

    __half * dB; 
    hipMalloc(&dB,  n * k * sizeof(__half));
    int ldb = k;

    __half * dC;
    hipMalloc(&dC, m * n * sizeof(__half));
    int ldc = m;

    __half * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(__half));

    hipMemcpy(dA, A, m * k * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(__half), hipMemcpyHostToDevice);
    
    float base;

    base = test_cublas_mm(m, n, k,  dA, lda, dB, ldb, dcheckC, ldc);
  
   
   
    hipMemcpy(C, dC ,m * n * sizeof(__half), hipMemcpyDeviceToHost);
    hipMemcpy(checkC, dcheckC, m * n * sizeof(__half), hipMemcpyDeviceToHost);
    //for (int i = 0; i < m * n; i++){
    // cout<<C[i]<<" ";	
    //}
    //check_C(C, m, n, checkC);

    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] checkC;

}



float test_cublas_mm(int m, int n, int k, 
         	     __half * dA, int lda, 
                     __half * dB, int ldb, 
                     __half * dC, int ldc){

    __half one = __float2half(1.0);
    __half zero = __float2half(0.0);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
        &one, dA, lda, dB, ldb, &zero, dC, ldc);
      check_cuda_error();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float real_time = milliseconds / 1000;

    cout <<"Runing time of culashgemm:" << real_time <<" s." << endl;
    return real_time;
}






























