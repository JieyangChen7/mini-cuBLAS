
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <climits>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#define LL 15 * 1024 
using namespace std;

__global__ void array_generator(double * A, int iteration, int access_per_iter) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  int L = gridDim.x * blockDim.x;
  for (int i = 0; i < iteration; i++) {
    double * nextA = A + L * access_per_iter;
    for (int j = 0; j < access_per_iter; j++) {
      *(A + L * j) = (unsigned long long int)( nextA + L * j );
    }
    A = nextA;
  }
}


// Kernel for 2048 threads / sm
__global__ void global_memory_2048(double * A, int iteration, int access_per_iter,
                              unsigned long long int * dStart, unsigned long long int * dEnd) {
  extern __shared__ double cache[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;

  //volatile clock_t start = 0;
  //volatile clock_t end = 0;
  //volatile unsigned long long sum_time = 0;

  double * a_next1;
  double * a_next2;
  double * a_next3;
  double * a_next4;
  double * a_next5;
  double * a_next6;
  double * a_next7;
  double * a_next8;
  
  double * a_next9;
  double * a_next10;
  double * a_next11;
  double * a_next12;
  
  double * a_curr1 = A;
  
  for (int i = 0; i < iteration; i++) {
    //start = clock();                                                                                                                      
    a_next1 = (double *)(unsigned long long int) *a_curr1;
    a_next2 = (double *)(unsigned long long int) *(a_curr1 + LL);
    
    a_next3 = (double *)(unsigned long long int) *(a_curr1 + LL * 2);
    a_next4 = (double *)(unsigned long long int) *(a_curr1 + LL * 3);
    
    a_next5 = (double *)(unsigned long long int) *(a_curr1 + LL * 4);
    a_next6 = (double *)(unsigned long long int) *(a_curr1 + LL * 5);
    a_next7 = (double *)(unsigned long long int) *(a_curr1 + LL * 6);
    a_next8 = (double *)(unsigned long long int) *(a_curr1 + LL * 7);
    
    a_next9 = (double *)(unsigned long long int) *(a_curr1 + LL * 8);
    a_next10 = (double *)(unsigned long long int) *(a_curr1 + LL * 9);
    a_next11 = (double *)(unsigned long long int) *(a_curr1 + LL * 10);
    a_next12 = (double *)(unsigned long long int) *(a_curr1 + LL * 11);
   
    __syncthreads();
    a_curr1 = a_next1;
  
    //end = clock(); 
  }
  
  *A += (unsigned long long int)a_next1;
  *A +=  (unsigned long long int)a_next2;
  *A +=  (unsigned long long int)a_next3;
  *A +=  (unsigned long long int)a_next4;
    
  *A +=  (unsigned long long int)a_next5;
  *A +=  (unsigned long long int)a_next6;
  *A +=  (unsigned long long int)a_next7;
  *A +=  (unsigned long long int)a_next8;
  
  *A +=  (unsigned long long int)a_next9;
  *A +=  (unsigned long long int)a_next10;
  *A +=  (unsigned long long int)a_next11;
  *A +=  (unsigned long long int)a_next12;
  
}

// Kernel for 1024 threads / sm
__global__ void global_memory_1024(double * A, int iteration, int access_per_iter,
                              unsigned long long int * dStart, unsigned long long int * dEnd) {
  extern __shared__ double cache[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;

  //volatile clock_t start = 0;
  //volatile clock_t end = 0;
  //volatile unsigned long long sum_time = 0;

  double * a_next1;
  double * a_next2;
  double * a_next3;
  double * a_next4;
  double * a_next5;
  double * a_next6;
  double * a_next7;
  double * a_next8;
  
  double * a_next9;
  double * a_next10;
  double * a_next11;
  double * a_next12;
  
  double * a_next13;
  double * a_next14;
  double * a_next15;
  double * a_next16;
  double * a_next17;
  double * a_next18;
  double * a_next19;
  double * a_next20;
  double * a_next21;
  double * a_next22;
  double * a_next23;
  double * a_next24;
  double * a_next25;
  
  double * a_curr1 = A;
  
  for (int i = 0; i < iteration; i++) {
    //start = clock();                                                                                                                      
    a_next1 = (double *)(unsigned long long int) *a_curr1;
    a_next2 = (double *)(unsigned long long int) *(a_curr1 + LL);
    
    a_next3 = (double *)(unsigned long long int) *(a_curr1 + LL * 2);
    a_next4 = (double *)(unsigned long long int) *(a_curr1 + LL * 3);
    
    a_next5 = (double *)(unsigned long long int) *(a_curr1 + LL * 4);
    a_next6 = (double *)(unsigned long long int) *(a_curr1 + LL * 5);
    a_next7 = (double *)(unsigned long long int) *(a_curr1 + LL * 6);
    a_next8 = (double *)(unsigned long long int) *(a_curr1 + LL * 7);
    
    a_next9 = (double *)(unsigned long long int) *(a_curr1 + LL * 8);
    a_next10 = (double *)(unsigned long long int) *(a_curr1 + LL * 9);
    a_next11 = (double *)(unsigned long long int) *(a_curr1 + LL * 10);
    a_next12 = (double *)(unsigned long long int) *(a_curr1 + LL * 11);
    
    a_next13 = (double *)(unsigned long long int) *(a_curr1 + LL * 12);
    a_next14 = (double *)(unsigned long long int) *(a_curr1 + LL * 13);
    a_next15 = (double *)(unsigned long long int) *(a_curr1 + LL * 14);
    a_next16 = (double *)(unsigned long long int) *(a_curr1 + LL * 15);
    
    a_next17 = (double *)(unsigned long long int) *(a_curr1 + LL * 16);
    a_next18 = (double *)(unsigned long long int) *(a_curr1 + LL * 17);
    a_next19 = (double *)(unsigned long long int) *(a_curr1 + LL * 18);
    a_next20 = (double *)(unsigned long long int) *(a_curr1 + LL * 19);
    a_next21 = (double *)(unsigned long long int) *(a_curr1 + LL * 20);
    a_next22 = (double *)(unsigned long long int) *(a_curr1 + LL * 21);
    a_next23 = (double *)(unsigned long long int) *(a_curr1 + LL * 22);
    a_next24 = (double *)(unsigned long long int) *(a_curr1 + LL * 23);
    a_next25 = (double *)(unsigned long long int) *(a_curr1 + LL * 24);
    
    __syncthreads();
    a_curr1 = a_next1;
    
    //end = clock(); 
  }
  
  *A += (unsigned long long int)a_next1;
  *A +=  (unsigned long long int)a_next2;
  *A +=  (unsigned long long int)a_next3;
  *A +=  (unsigned long long int)a_next4;
    
  *A +=  (unsigned long long int)a_next5;
  *A +=  (unsigned long long int)a_next6;
  *A +=  (unsigned long long int)a_next7;
  *A +=  (unsigned long long int)a_next8;
  
  *A +=  (unsigned long long int)a_next9;
  *A +=  (unsigned long long int)a_next10;
  *A +=  (unsigned long long int)a_next11;
  *A +=  (unsigned long long int)a_next12;

  *A +=  (unsigned long long int)a_next13;
  *A +=  (unsigned long long int)a_next14;
  *A +=  (unsigned long long int)a_next15;
  *A +=  (unsigned long long int)a_next16;

  *A +=  (unsigned long long int)a_next17;
  *A +=  (unsigned long long int)a_next18;
  *A +=  (unsigned long long int)a_next19;
  *A +=  (unsigned long long int)a_next20;

  *A +=  (unsigned long long int)a_next21;
  *A +=  (unsigned long long int)a_next22;
  *A +=  (unsigned long long int)a_next23;
  *A +=  (unsigned long long int)a_next24;
  *A +=  (unsigned long long int)a_next25;
  
}



void test_2048(int block_size){
  int iteration = 1000;
  int access_per_iter = 12;
  int SM = 15;
  int block_per_sm = 1024/block_size;
  int total_block = SM * block_per_sm;
  //int block_size = 1024;

  int n = total_block * block_size * access_per_iter * (iteration + 1);
  double * A = new double[n];
  unsigned long long int * start = new unsigned long long int[n];
  unsigned long long int * end = new unsigned long long int[n];
  unsigned long long int * dStart;
  unsigned long long int * dEnd;
  double * dA;
  hipMalloc(&dA, (n) * sizeof(double));
  hipMalloc((void**)&dStart, n * sizeof(unsigned long long int));
  hipMalloc((void**)&dEnd, n * sizeof(unsigned long long int));

  array_generator<<<total_block, block_size>>>(dA, iteration, access_per_iter);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("<array_gene>Error: %s\n", hipGetErrorString(err));

  clock_t t = clock();
  global_memory_1024<<<total_block, block_size, 49152 / block_per_sm>>>(dA, iteration, access_per_iter, dStart, dEnd);
  hipDeviceSynchronize();
  t = clock() - t;

  float real_time = ((float)t)/CLOCKS_PER_SEC;
  cout <<"Runing time: " << real_time << " s." << endl;
  long long total_byte = total_block * block_size * sizeof(double) * access_per_iter;
  double total_gb = total_byte/1e9;
  total_gb *= iteration;
  cout << "Total data requested:"<<total_gb << " GB."<< endl;
  double throughput = total_gb/real_time;
  cout <<"Throughput: " << throughput << " GB/s." << endl;
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("<global_memory>Error: %s\n", hipGetErrorString(err));

  hipMemcpy(A, dA, n * sizeof(double), hipMemcpyDeviceToHost);

}

void test_1024(int block_size){
  int iteration = 1000;
  int access_per_iter = 25;
  int SM = 15;
  int block_per_sm = 1024/block_size;
  int total_block = SM * block_per_sm;
  //int block_size = 1024;

  int n = total_block * block_size * access_per_iter * (iteration + 1);
  double * A = new double[n];
  unsigned long long int * start = new unsigned long long int[n];
  unsigned long long int * end = new unsigned long long int[n];
  unsigned long long int * dStart;
  unsigned long long int * dEnd;
  double * dA;
  hipMalloc(&dA, (n) * sizeof(double));
  hipMalloc((void**)&dStart, n * sizeof(unsigned long long int));
  hipMalloc((void**)&dEnd, n * sizeof(unsigned long long int));

  array_generator<<<total_block, block_size>>>(dA, iteration, access_per_iter);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("<array_gene>Error: %s\n", hipGetErrorString(err));

  clock_t t = clock();
  global_memory_1024<<<total_block, block_size, 49152 / block_per_sm>>>(dA, iteration, access_per_iter, dStart, dEnd);
  hipDeviceSynchronize();
  t = clock() - t;

  float real_time = ((float)t)/CLOCKS_PER_SEC;
  cout <<"Runing time: " << real_time << " s." << endl;
  long long total_byte = total_block * block_size * sizeof(double) * access_per_iter;
  double total_gb = total_byte/1e9;
  total_gb *= iteration;
  cout << "Total data requested:"<<total_gb << " GB."<< endl;
  double throughput = total_gb/real_time;
  cout <<"Throughput: " << throughput << " GB/s." << endl;
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("<global_memory>Error: %s\n", hipGetErrorString(err));

  hipMemcpy(A, dA, n * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dA);
  hipFree(dStart);
  hipFree(dEnd);
  delete [] A;
  delete [] start;
  delete [] end;  
}


int main(){
  //int i = 1024;
  for (int i = 64; i <= 1024; i *= 2) {
    cout << "block size: " << i << endl;
    test_1024(i);
  }

}
