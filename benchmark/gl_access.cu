
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <climits>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#define LL 15 * 512 
using namespace std;

__global__ void array_generator(double * A, int iteration, int access_per_iter) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  int L = gridDim.x * blockDim.x;
  for (int i = 0; i < iteration; i++) {
    double * nextA = A + L * access_per_iter;
    for (int j = 0; j < access_per_iter; j++) {
      *(A + L * j) = (unsigned long long int)( nextA + L * j );
    }
    A = nextA;
  }
}


// Kernel for 2048 threads / sm
// Max register use is: 32
// this version disable unroll
__global__ void global_memory_2048(double * A, int iteration, int access_per_iter,
                              unsigned long long int * dStart, unsigned long long int * dEnd) {
  extern __shared__ double cache[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;

  //volatile clock_t start = 0;
  //volatile clock_t end = 0;
  //volatile unsigned long long sum_time = 0;

  double * a_next1 = A;
  double * a_next2 = A + LL;
  double * a_next3 = A + LL * 2;
  double * a_next4 = A + LL * 3;
  double * a_next5 = A + LL * 4;
  double * a_next6 = A + LL * 5;
  double * a_next7 = A + LL * 6;

  # pragma unroll 1
  for (int i = 0; i < iteration; i++) {
    //start = clock();                                                                                                                      
    a_next1 = (double *)(unsigned long long int) *a_next1;
    a_next2 = (double *)(unsigned long long int) *a_next2;
    
    a_next3 = (double *)(unsigned long long int) *a_next3;
    a_next4 = (double *)(unsigned long long int) *a_next4;
    
    a_next5 = (double *)(unsigned long long int) *a_next5;
    a_next6 = (double *)(unsigned long long int) *a_next6;
    a_next7 = (double *)(unsigned long long int) *a_next7;

    //end = clock(); 
  }
  
  *A += (unsigned long long int)a_next1;
  *A +=  (unsigned long long int)a_next2;
  *A +=  (unsigned long long int)a_next3;
  *A +=  (unsigned long long int)a_next4;
    
  *A +=  (unsigned long long int)a_next5;
  *A +=  (unsigned long long int)a_next6;
  *A +=  (unsigned long long int)a_next7;

}

// Kernel for 1024 threads / sm
// Max register use is 64
// this version disable unroll
__global__ void global_memory_1024(double * A, int iteration, int access_per_iter,
                              unsigned long long int * dStart, unsigned long long int * dEnd) {
  extern __shared__ double cache[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;

  //volatile clock_t start = 0;
  //volatile clock_t end = 0;
  //volatile unsigned long long sum_time = 0;

  double * a_next1 = A;
  double * a_next2 = A + LL;
  double * a_next3 = A + LL * 2;
  double * a_next4 = A + LL * 3;
  double * a_next5 = A + LL * 4;
  double * a_next6 = A + LL * 5;
  double * a_next7 = A + LL * 6;
  double * a_next8 = A + LL * 7;
  
  double * a_next9 = A + LL * 8;
  double * a_next10 = A + LL * 9;
  double * a_next11 = A + LL * 10;
  double * a_next12 = A + LL * 11;
  
  double * a_next13 = A + LL * 12;
  double * a_next14 = A + LL * 13;
  double * a_next15 = A + LL * 14;
  double * a_next16 = A + LL * 15;
  double * a_next17 = A + LL * 16;
  double * a_next18 = A + LL * 17;
  double * a_next19 = A + LL * 18;
  double * a_next20 = A + LL * 19;
  double * a_next21 = A + LL * 20;
  double * a_next22 = A + LL * 21;
  double * a_next23 = A + LL * 22;
   
# pragma unroll 1
  for (int i = 0; i < iteration; i++) {
    //start = clock();                                                                                                                      
    a_next1 = (double *)(unsigned long long int) *a_next1;
    a_next2 = (double *)(unsigned long long int) *a_next2; 
    a_next3 = (double *)(unsigned long long int) *a_next3;
    a_next4 = (double *)(unsigned long long int) *a_next4;
    
    a_next5 = (double *)(unsigned long long int) *a_next5;
    a_next6 = (double *)(unsigned long long int) *a_next6;
    a_next7 = (double *)(unsigned long long int) *a_next7;
    a_next8 = (double *)(unsigned long long int) *a_next8;
    
    a_next9 = (double *)(unsigned long long int) *a_next9;
    a_next10 = (double *)(unsigned long long int) *a_next10;
    a_next11 = (double *)(unsigned long long int) *a_next11;
    a_next12 = (double *)(unsigned long long int) *a_next12;
    
    a_next13 = (double *)(unsigned long long int) *a_next13;
    a_next14 = (double *)(unsigned long long int) *a_next14;
    a_next15 = (double *)(unsigned long long int) *a_next15;
    a_next16 = (double *)(unsigned long long int) *a_next16;
    
    a_next17 = (double *)(unsigned long long int) *a_next17;
    a_next18 = (double *)(unsigned long long int) *a_next18;
    a_next19 = (double *)(unsigned long long int) *a_next19;
    a_next20 = (double *)(unsigned long long int) *a_next20;
    
    a_next21 = (double *)(unsigned long long int) *a_next21;
    a_next22 = (double *)(unsigned long long int) *a_next22;
    a_next23 = (double *)(unsigned long long int) *a_next23;
    
    //end = clock(); 
  }
  
  *A += (unsigned long long int)a_next1;
  *A +=  (unsigned long long int)a_next2;
  *A +=  (unsigned long long int)a_next3;
  *A +=  (unsigned long long int)a_next4;
    
  *A +=  (unsigned long long int)a_next5;
  *A +=  (unsigned long long int)a_next6;
  *A +=  (unsigned long long int)a_next7;
  *A +=  (unsigned long long int)a_next8;
  
  *A +=  (unsigned long long int)a_next9;
  *A +=  (unsigned long long int)a_next10;
  *A +=  (unsigned long long int)a_next11;
  *A +=  (unsigned long long int)a_next12;

  *A +=  (unsigned long long int)a_next13;
  *A +=  (unsigned long long int)a_next14;
  *A +=  (unsigned long long int)a_next15;
  *A +=  (unsigned long long int)a_next16;

  *A +=  (unsigned long long int)a_next17;
  *A +=  (unsigned long long int)a_next18;
  *A +=  (unsigned long long int)a_next19;
  *A +=  (unsigned long long int)a_next20;

  *A +=  (unsigned long long int)a_next21;
  *A +=  (unsigned long long int)a_next22;
  *A +=  (unsigned long long int)a_next23;  
}




// Kernel for 1024 threads / sm
// Max regiter use js 64
// this version let compilter to automatic unroll
__global__ void global_memory_1024_2(double * A, int iteration, int access_per_iter,
                              unsigned long long int * dStart, unsigned long long int * dEnd) {
  extern __shared__ double cache[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;

  //volatile clock_t start = 0;
  //volatile clock_t end = 0;
  //volatile unsigned long long sum_time = 0;

  double * a_next1 = A;
  double * a_next2 = A + LL;
  double * a_next3 = A + LL * 2;
  double * a_next4 = A + LL * 3;

  double * a_next5 = A + LL * 4;
  double * a_next6 = A + LL * 5;
  double * a_next7 = A + LL * 6;
  double * a_next8 = A + LL * 7;

  double * a_next9 = A + LL * 8;
  double * a_next10 = A + LL * 9;
  double * a_next11 = A + LL * 10;
  double * a_next12 = A + LL * 11;

  double * a_next13 = A + LL * 12;
  double * a_next14 = A + LL * 13;

  
  for (int i = 0; i < iteration; i++) {
    //start = clock();                                                                                                                      
    a_next1 = (double *)(unsigned long long int) *a_next1;
    a_next2 = (double *)(unsigned long long int) *a_next2;  
    a_next3 = (double *)(unsigned long long int) *a_next3;
    a_next4 = (double *)(unsigned long long int) *a_next4;
    
    a_next5 = (double *)(unsigned long long int) *a_next5;
    a_next6 = (double *)(unsigned long long int) *a_next6;
    a_next7 = (double *)(unsigned long long int) *a_next7;
    a_next8 = (double *)(unsigned long long int) *a_next8;
    
    a_next9 = (double *)(unsigned long long int) *a_next9;
    a_next10 = (double *)(unsigned long long int) *a_next10;
    a_next11 = (double *)(unsigned long long int) *a_next11;
    a_next12 = (double *)(unsigned long long int) *a_next12;
    
    a_next13 = (double *)(unsigned long long int) *a_next13;
    a_next14 = (double *)(unsigned long long int) *a_next14;

    //end = clock(); 
  }
  
  *A += (unsigned long long int)a_next1;
  *A +=  (unsigned long long int)a_next2;
  *A +=  (unsigned long long int)a_next3;
  *A +=  (unsigned long long int)a_next4;
    
  *A +=  (unsigned long long int)a_next5;
  *A +=  (unsigned long long int)a_next6;
  *A +=  (unsigned long long int)a_next7;
  *A +=  (unsigned long long int)a_next8;
  
  *A +=  (unsigned long long int)a_next9;
  *A +=  (unsigned long long int)a_next10;
  *A +=  (unsigned long long int)a_next11;
  *A +=  (unsigned long long int)a_next12;

  *A +=  (unsigned long long int)a_next13;
  *A +=  (unsigned long long int)a_next14;
}



// Kernel for 512 threads / sm
// Max register use is 128
// this version disable unroll
__global__ void global_memory_512(double * A, int iteration, int access_per_iter,
                              unsigned long long int * dStart, unsigned long long int * dEnd) {
  extern __shared__ double cache[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;

  //volatile clock_t start = 0;
  //volatile clock_t end = 0;
  //volatile unsigned long long sum_time = 0;

  double * a_next1 = A;
  double * a_next2 = A + LL;
  double * a_next3 = A + LL * 2;
  double * a_next4 = A + LL * 3;
  double * a_next5 = A + LL * 4;
  double * a_next6 = A + LL * 5;
  double * a_next7 = A + LL * 6;
  double * a_next8 = A + LL * 7;
  
  double * a_next9 = A + LL * 8;
  double * a_next10 = A + LL * 9;
  double * a_next11 = A + LL * 10;
  double * a_next12 = A + LL * 11;
  
  double * a_next13 = A + LL * 12;
  double * a_next14 = A + LL * 13;
  double * a_next15 = A + LL * 14;
  double * a_next16 = A + LL * 15;
  double * a_next17 = A + LL * 16;
  double * a_next18 = A + LL * 17;
  double * a_next19 = A + LL * 18;
  double * a_next20 = A + LL * 19;
  double * a_next21 = A + LL * 20;
  double * a_next22 = A + LL * 21;
  double * a_next23 = A + LL * 22;
  double * a_next24 = A + LL * 23;
  double * a_next25 = A + LL * 24;
  double * a_next26 = A + LL * 25;
  double * a_next27 = A + LL * 26;
  double * a_next28 = A + LL * 27;
  double * a_next29 = A + LL * 28; 
  double * a_next30 = A + LL * 29;

  double * a_next31 = A + LL * 31;
  double * a_next32 = A + LL * 32;
  double * a_next33 = A + LL * 33; 
  double * a_next34 = A + LL * 34;
  double * a_next35 = A + LL * 35; 
  double * a_next36 = A + LL * 36;

  double * a_next37 = A + LL * 37; 
  double * a_next38 = A + LL * 38;
  double * a_next39 = A + LL * 39; 
  double * a_next40 = A + LL * 40;

  double * a_next41 = A + LL * 41;
  double * a_next42 = A + LL * 42;
  double * a_next43 = A + LL * 43; 
  double * a_next44 = A + LL * 44;
  double * a_next45 = A + LL * 45; 
  double * a_next46 = A + LL * 46;

  double * a_next47 = A + LL * 47; 
  double * a_next48 = A + LL * 48;
  double * a_next49 = A + LL * 49; 
  double * a_next50 = A + LL * 50;

  double * a_next51 = A + LL * 51;
  double * a_next52 = A + LL * 52; 
  double * a_next53 = A + LL * 53;
  double * a_next54 = A + LL * 54;
  double * a_next55 = A + LL * 55;


# pragma unroll 1
  for (int i = 0; i < iteration; i++) {
    //start = clock();                                                                                                                      
    a_next1 = (double *)(unsigned long long int) *a_next1;
    a_next2 = (double *)(unsigned long long int) *a_next2; 
    a_next3 = (double *)(unsigned long long int) *a_next3;
    a_next4 = (double *)(unsigned long long int) *a_next4;
    
    a_next5 = (double *)(unsigned long long int) *a_next5;
    a_next6 = (double *)(unsigned long long int) *a_next6;
    a_next7 = (double *)(unsigned long long int) *a_next7;
    a_next8 = (double *)(unsigned long long int) *a_next8;
    
    a_next9 = (double *)(unsigned long long int) *a_next9;
    a_next10 = (double *)(unsigned long long int) *a_next10;
    a_next11 = (double *)(unsigned long long int) *a_next11;
    a_next12 = (double *)(unsigned long long int) *a_next12;
    
    a_next13 = (double *)(unsigned long long int) *a_next13;
    a_next14 = (double *)(unsigned long long int) *a_next14;
    a_next15 = (double *)(unsigned long long int) *a_next15;
    a_next16 = (double *)(unsigned long long int) *a_next16;
    
    a_next17 = (double *)(unsigned long long int) *a_next17;
    a_next18 = (double *)(unsigned long long int) *a_next18;
    a_next19 = (double *)(unsigned long long int) *a_next19;
    a_next20 = (double *)(unsigned long long int) *a_next20;
    
    a_next21 = (double *)(unsigned long long int) *a_next21;
    a_next22 = (double *)(unsigned long long int) *a_next22;
    a_next23 = (double *)(unsigned long long int) *a_next23;
    a_next24 = (double *)(unsigned long long int) *a_next24;

    a_next25 = (double *)(unsigned long long int) *a_next25;
    a_next26 = (double *)(unsigned long long int) *a_next26;
    a_next27 = (double *)(unsigned long long int) *a_next27;
    a_next28 = (double *)(unsigned long long int) *a_next28;

    a_next29 = (double *)(unsigned long long int) *a_next29;
    a_next30 = (double *)(unsigned long long int) *a_next30;
    a_next31 = (double *)(unsigned long long int) *a_next31;
    a_next32 = (double *)(unsigned long long int) *a_next32;

    a_next33 = (double *)(unsigned long long int) *a_next33;
    a_next34 = (double *)(unsigned long long int) *a_next34;
    a_next35 = (double *)(unsigned long long int) *a_next35;
    a_next36 = (double *)(unsigned long long int) *a_next36;

    a_next37 = (double *)(unsigned long long int) *a_next37;
    a_next38 = (double *)(unsigned long long int) *a_next38;
    a_next39 = (double *)(unsigned long long int) *a_next39;
    a_next40 = (double *)(unsigned long long int) *a_next40;

    a_next41 = (double *)(unsigned long long int) *a_next41;
    a_next42 = (double *)(unsigned long long int) *a_next42;
    a_next43 = (double *)(unsigned long long int) *a_next43;
    a_next44 = (double *)(unsigned long long int) *a_next44;
    
    a_next45 = (double *)(unsigned long long int) *a_next45;
    a_next46 = (double *)(unsigned long long int) *a_next46;
    a_next47 = (double *)(unsigned long long int) *a_next47;
    a_next48 = (double *)(unsigned long long int) *a_next48;
    
    a_next49 = (double *)(unsigned long long int) *a_next49;
    a_next50 = (double *)(unsigned long long int) *a_next50;
    a_next51 = (double *)(unsigned long long int) *a_next51;
    a_next52 = (double *)(unsigned long long int) *a_next52;
    
    a_next53 = (double *)(unsigned long long int) *a_next53;
    a_next54 = (double *)(unsigned long long int) *a_next54;
    a_next55 = (double *)(unsigned long long int) *a_next55;
    
    //end = clock(); 
  }
  
  *A += (unsigned long long int)a_next1;
  *A +=  (unsigned long long int)a_next2;
  *A +=  (unsigned long long int)a_next3;
  *A +=  (unsigned long long int)a_next4;
    
  *A +=  (unsigned long long int)a_next5;
  *A +=  (unsigned long long int)a_next6;
  *A +=  (unsigned long long int)a_next7;
  *A +=  (unsigned long long int)a_next8;
  
  *A +=  (unsigned long long int)a_next9;
  *A +=  (unsigned long long int)a_next10;
  *A +=  (unsigned long long int)a_next11;
  *A +=  (unsigned long long int)a_next12;

  *A +=  (unsigned long long int)a_next13;
  *A +=  (unsigned long long int)a_next14;
  *A +=  (unsigned long long int)a_next15;
  *A +=  (unsigned long long int)a_next16;

  *A +=  (unsigned long long int)a_next17;
  *A +=  (unsigned long long int)a_next18;
  *A +=  (unsigned long long int)a_next19;
  *A +=  (unsigned long long int)a_next20;

  *A +=  (unsigned long long int)a_next21;
  *A +=  (unsigned long long int)a_next22;
  *A +=  (unsigned long long int)a_next23;
  *A +=  (unsigned long long int)a_next24;

  *A +=  (unsigned long long int)a_next25;
  *A +=  (unsigned long long int)a_next26;
  *A +=  (unsigned long long int)a_next27;
  *A +=  (unsigned long long int)a_next28;

  *A +=  (unsigned long long int)a_next29;
  *A +=  (unsigned long long int)a_next30;
  *A +=  (unsigned long long int)a_next31;
  *A +=  (unsigned long long int)a_next32;

  *A +=  (unsigned long long int)a_next33;
  *A +=  (unsigned long long int)a_next34;
  *A +=  (unsigned long long int)a_next35;
  *A +=  (unsigned long long int)a_next36;

  *A +=  (unsigned long long int)a_next37;
  *A +=  (unsigned long long int)a_next38;
  *A +=  (unsigned long long int)a_next39;
  *A +=  (unsigned long long int)a_next40;

  *A +=  (unsigned long long int)a_next41;
  *A +=  (unsigned long long int)a_next42;
  *A +=  (unsigned long long int)a_next43;
  *A +=  (unsigned long long int)a_next44;

  *A +=  (unsigned long long int)a_next45;
  *A +=  (unsigned long long int)a_next46;
  *A +=  (unsigned long long int)a_next47;
  *A +=  (unsigned long long int)a_next48;

  *A +=  (unsigned long long int)a_next49;
  *A +=  (unsigned long long int)a_next50;
  *A +=  (unsigned long long int)a_next51;
  *A +=  (unsigned long long int)a_next52;

  *A +=  (unsigned long long int)a_next53;
  *A +=  (unsigned long long int)a_next54;
  *A +=  (unsigned long long int)a_next55;

}


void test_2048(int block_size){
  int iteration = 1000;
  int access_per_iter = 7;
  int SM = 15;
  int block_per_sm = 2048/block_size;
  int total_block = SM * block_per_sm;
  //int block_size = 1024;
  cout << "Total concurrent threads/SM: " << block_per_sm * block_size << endl;
  int n = total_block * block_size * access_per_iter * (iteration + 1);
  double * A = new double[n];
  unsigned long long int * start = new unsigned long long int[n];
  unsigned long long int * end = new unsigned long long int[n];
  unsigned long long int * dStart;
  unsigned long long int * dEnd;
  double * dA;
  hipMalloc(&dA, (n) * sizeof(double));
  hipMalloc((void**)&dStart, n * sizeof(unsigned long long int));
  hipMalloc((void**)&dEnd, n * sizeof(unsigned long long int));

  array_generator<<<total_block, block_size>>>(dA, iteration, access_per_iter);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("<array_gene>Error: %s\n", hipGetErrorString(err));

  clock_t t = clock();
  global_memory_2048<<<total_block, block_size, 49152 / block_per_sm>>>(dA, iteration, access_per_iter, dStart, dEnd);
  hipDeviceSynchronize();
  t = clock() - t;

  float real_time = ((float)t)/CLOCKS_PER_SEC;
  cout <<"Runing time: " << real_time << " s." << endl;
  long long total_byte = total_block * block_size * sizeof(double) * access_per_iter;
  double total_gb = total_byte/1e9;
  total_gb *= iteration;
  cout << "Total data requested:"<<total_gb << " GB."<< endl;
  double throughput = total_gb/real_time;
  cout <<"Throughput: " << throughput << " GB/s." << endl;
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("<global_memory>Error: %s\n", hipGetErrorString(err));

  hipMemcpy(A, dA, n * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dA);
  hipFree(dStart);
  hipFree(dEnd);
  delete [] A;
  delete [] start;
  delete [] end;  

}

void test_1024(int block_size){
  int iteration = 1000;
  int access_per_iter = 23;
  int SM = 15;
  int block_per_sm = 1024/block_size;
  int total_block = SM * block_per_sm;
  //int block_size = 1024;

  int n = total_block * block_size * access_per_iter * (iteration + 1);
  double * A = new double[n];
  unsigned long long int * start = new unsigned long long int[n];
  unsigned long long int * end = new unsigned long long int[n];
  unsigned long long int * dStart;
  unsigned long long int * dEnd;
  double * dA;
  hipMalloc(&dA, (n) * sizeof(double));
  hipMalloc((void**)&dStart, n * sizeof(unsigned long long int));
  hipMalloc((void**)&dEnd, n * sizeof(unsigned long long int));

  array_generator<<<total_block, block_size>>>(dA, iteration, access_per_iter);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("<array_gene>Error: %s\n", hipGetErrorString(err));

  clock_t t = clock();
  global_memory_1024<<<total_block, block_size, 49152 / block_per_sm>>>(dA, iteration, access_per_iter, dStart, dEnd);
  hipDeviceSynchronize();
  t = clock() - t;

  float real_time = ((float)t)/CLOCKS_PER_SEC;
  cout <<"Runing time: " << real_time << " s." << endl;
  long long total_byte = total_block * block_size * sizeof(double) * access_per_iter;
  double total_gb = total_byte/1e9;
  total_gb *= iteration;
  cout << "Total data requested:"<<total_gb << " GB."<< endl;
  double throughput = total_gb/real_time;
  cout <<"Throughput: " << throughput << " GB/s." << endl;
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("<global_memory>Error: %s\n", hipGetErrorString(err));

  hipMemcpy(A, dA, n * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dA);
  hipFree(dStart);
  hipFree(dEnd);
  delete [] A;
  delete [] start;
  delete [] end;  
}


void test_512(int block_size){
  int iteration = 1000;
  int access_per_iter = 34;
  int SM = 15;
  int block_per_sm = 512/block_size;
  int total_block = SM * block_per_sm;
  //int block_size = 1024;

  int n = total_block * block_size * access_per_iter * (iteration + 1);
  double * A = new double[n];
  unsigned long long int * start = new unsigned long long int[n];
  unsigned long long int * end = new unsigned long long int[n];
  unsigned long long int * dStart;
  unsigned long long int * dEnd;
  double * dA;
  hipMalloc(&dA, (n) * sizeof(double));
  hipMalloc((void**)&dStart, n * sizeof(unsigned long long int));
  hipMalloc((void**)&dEnd, n * sizeof(unsigned long long int));

  array_generator<<<total_block, block_size>>>(dA, iteration, access_per_iter);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("<array_gene>Error: %s\n", hipGetErrorString(err));

  clock_t t = clock();
  global_memory_512<<<total_block, block_size, 49152 / block_per_sm>>>(dA, iteration, access_per_iter, dStart, dEnd);
  hipDeviceSynchronize();
  t = clock() - t;

  float real_time = ((float)t)/CLOCKS_PER_SEC;
  cout <<"Runing time: " << real_time << " s." << endl;
  long long total_byte = total_block * block_size * sizeof(double) * access_per_iter;
  double total_gb = total_byte/1e9;
  total_gb *= iteration;
  cout << "Total data requested:"<<total_gb << " GB."<< endl;
  double throughput = total_gb/real_time;
  cout <<"Throughput: " << throughput << " GB/s." << endl;
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("<global_memory>Error: %s\n", hipGetErrorString(err));

  hipMemcpy(A, dA, n * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dA);
  hipFree(dStart);
  hipFree(dEnd);
  delete [] A;
  delete [] start;
  delete [] end;  
}


int main(){
  if (LL / 15 == 1024) { 
    for (int i = 64; i <= 1024; i *= 2) {
      cout << "block size: " << i << endl;
      test_1024(i);
    }
  } else if (LL / 15 == 2048) {
    for (int i = 128; i <= 1024; i *= 2) {
      cout << "block size: " << i << endl;
      test_2048(i);
    }
  } else if (LL / 15 == 512) {
    for (int i = 32; i <= 1024; i *= 2) {
      cout << "block size: " << i << endl;
      test_512(i);
    }
  }
  

}
