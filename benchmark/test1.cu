
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;

__global__ void array_generator(int n, double * A) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //clock_t start = clock();
  A[idx] = (unsigned long long int)(A + idx + blockDim.x);
  //clock_t end = clock();
  //printf("%d\n", end-start);
}

__global__ void global_memory(int n, double * A, int space, int iteration, unsigned long long int * T) {
  int idx = blockIdx.x * space + threadIdx.x;
  A = A + idx;
  volatile clock_t start = 0;
  volatile clock_t end = 0;
  volatile unsigned long long sum_time = 0;

  for (int i = 0; i < iteration; i++) {
    start = clock();
  	A = (double *)(unsigned long long int) *A;
    end = clock();
    sum_time += (end - start);
  }
  T[idx] = sum_time;

  //printf("%d ", end-start);
  //printf("SE: %d %d", start, end);

}

__global__ void tid_time(int iteration, unsigned long long int * T) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
   register int start = 0;
   register int end = 0;
  unsigned long long sum_time = 0;
   register int idx2 = 0;
   register int a = 1;
   register int b = 2;
   register int c = 3;
  for (int i = 0; i < iteration; i++) {
    //start = clock();
    //idx2 = a * b + c;
    // asm volatile ("mov.u32 %0, %%clock;" : "=r"(start));

    // asm volatile ("mad.lo.s32 %0, %1, %2, %3;" : "=r"(idx2) : "r"(a), "r"(b), "r"(c));
    // asm volatile ("mov.u32 %0, %%clock;" : "=r"(end));

    asm volatile ("{\n\t"
                  "mov.u32 %0, %%clock;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mad.lo.s32 %1, %3, %4, %1;\n\t"
                  "mov.u32 %2, %%clock;\n\t"
                  "}"
                  :  "=r"(start), "=r"(idx2), "=r"(end): "r"(a), "r"(b), "r"(c) : "memory"
                  );

    
    //end = clock();
    sum_time += (end - start);
  }
  printf("%d", idx2);
  T[idx] = sum_time + idx2;

  //printf("%d ", end-start);
  //printf("SE: %d %d", start, end);

}



int main(){
  int n = 128;
  int B = 16;
  double * A = new double[n + B];
  unsigned long long int * T = new unsigned long long int[n];
  
  double * dA;
  unsigned long long int *dT;
  hipMalloc(&dA, (n + B) * sizeof(double));
  hipMalloc((void**)&dT, n * sizeof(unsigned long long int));

  //array_generator<<<n/B, B>>>(n, dA);
  //global_memory<<<n/B, B>>>(n, dA, B, 1, dT);
  tid_time<<<n/B, B>>>(1, dT);
  hipMemcpy(A, dA, (n + B) * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(T, dT, n * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
 
//  for (int i = 0; i < n + B; i++) {
//  	cout << A[i] << " ";
//  }
    for (int i = 0; i < n; i++) {
    	cout << "" << i << " "<< T[i] << endl;;
    }
}
