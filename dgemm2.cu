
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>
#include <time.h>
#include <stdio.h>
#define TEST_RUN 50 
#define ESP 10e-10
using namespace std;

__global__ void
dgemm_kernel2(int m, int n, int k, 
              double * A, int lda, 
              double * B, int ldb, 
              double * C, int ldc);

__global__ void
dgemm_kernel2_1(int m, int n, int k, 
                double * A, int lda, 
                double * B, int ldb, 
                double * C, int ldc);

__global__ void
dgemm_kernel3(int m, int n, int k, int T, 
              double * A, int lda, 
              double * B, int ldb, 
              double * C, int ldc);

__global__ void
dgemm_kernel4(int m, int n, int k, int T, 
              double * A, int lda, 
              double * B, int ldb, 
              double * C, int ldc);

__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, int t,
                double * A, int lda, 
                double * B, int ldb, 
                double * C, int ldc);

__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t,
                double * A, int lda, 
                double * B, int ldb, 
                double * C, int ldc);

void check_C(double * dC, int m, int n, double * checkC);

void test_cublas_mm(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc);

void test_kernel2(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc);

void test_kernel2_1(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc);

void test_kernel3(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc);

void test_kernel4(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc);

void test_kernel4_1(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc);

void test_kernel4_2(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc);

void test(int m, int k);

int main(){
  for (int i = 128; i <= 32768; i *= 2){
    i = 20480;
    cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 1 << ")" << endl;
    test(i, i);
  }
}

void test(int m, int k){
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 2;
    //int k = 20480;
    double * A = new double[m * k];
    double * B = new double[n * k];
    double * C = new double[m * n];
    double * checkC = new double[m * n];     

    for (int i = 0; i < m * k; i++){
    	A[i] = i;
    }

    //    for (int i = 0; i < m; i++){
    // for (int j = 0; j < k; j++){
    //	cout << *( A + i + j * m) << " ";
    // }
    // cout << endl;
    //}
    
    for (int i = 0; i < n * k; i++){
    	B[i] = 1;
    }
    
    double * dA;
    hipMalloc(&dA, m * k * sizeof(double));
    int lda = m;

    double * dB; 
    hipMalloc(&dB,  n * k * sizeof(double));
    int ldb = k;

    double * dC;
    hipMalloc(&dC, m * n * sizeof(double));
    int ldc = m;

    double * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(double));

    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    
    test_cublas_mm(m, n, k,  dA, lda, dB, ldb, dcheckC, ldc);
    test_kernel2(m, n, k, dA, lda, dB, ldb, dC, ldc);
    test_kernel2_1(m, n, k, dA, lda, dB, ldb, dC, ldc);
    test_kernel3(m, n, k, dA, lda, dB, ldb, dC, ldc);
    test_kernel4(m, n, k, dA, lda, dB, ldb, dC, ldc);
    test_kernel4_1(m, n, k, dA, lda, dB, ldb, dC, ldc);
    test_kernel4_2(m, n, k, dA, lda, dB, ldb, dC, ldc);
    
   
    hipMemcpy(C, dC ,m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(checkC, dcheckC, m * n * sizeof(double), hipMemcpyDeviceToHost);
    //for (int i = 0; i < m * n; i++){
    // cout<<C[i]<<" ";	
    //}
    //check_C(C, m, n, checkC);

    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    delete[] checkC;

}



void test_cublas_mm(int m, int n, int k, 
         double * dA, int lda, 
         double * dB, int ldb, 
         double * dC, int ldc){

    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
        &one, dA, lda, dB, ldb, &zero, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of culasdgemm:" << real_time <<" ms." << endl;
}

void test_kernel2(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc){


    int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();

    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, 
              dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of dgemm_kernel2: " << real_time << " ms." << endl;    

} 


void test_kernel2_1(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc){
  


  int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock(); 
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2_1<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
                  dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of dgemm_kernel2_1: " << real_time << " ms." << endl;


}

void test_kernel3(int m, int n, int k, 
          double * dA, int lda, 
          double * dB, int ldb, 
          double * dC, int ldc){

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel3<<<blocksPerGrid, threadsPerBlock,  T * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    hipDeviceSynchronize();
    t = clock() - t;

    float real_time = ((float)t)/CLOCKS_PER_SEC;
    cout <<"Runing time of dgemm_kernel3: " << real_time << " ms." << endl;     
}


void test_kernel4(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc){

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4<<<blocksPerGrid, threadsPerBlock, ((T) + (T * T)) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    }
    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;
    cout <<"Runing time of dgemm_kernel4: " << real_time << " ms." << endl;    
}

void test_kernel4_1(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_1<<<blocksPerGrid, threadsPerBlock, ((T * 2) + (T * tt)) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
    

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;


    cout <<"Runing time of dgemm_kernel4_1: " << real_time << " ms." << endl;   

}


void test_kernel4_2(int m, int n, int k, 
            double * dA, int lda, 
            double * dB, int ldb, 
            double * dC, int ldc){    
    int T = 64;
    int tt = 4;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_1<<<blocksPerGrid, threadsPerBlock, ((T * 2)) * sizeof(double)>>>(m, n, k, T, tt, dA, lda, dB, ldb, dC, ldc);
    

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;


    cout <<"Runing time of dgemm_kernel4_1: " << real_time << " ms." << endl;   

}

void check_C(double * dC, int m, int n, double * checkC) {
  for (int i = 0; i < m * n; i++){
    //cout << i << endl;
    if (fabs(dC[i] - checkC[i]) > ESP){
      cout << "error:" << fabs(dC[i] - checkC[i]) << endl;
      return;
    }
  }
  cout << "correct" << endl;
}


__global__ void
dgemm_kernel2(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
	//determine the row to process
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	A = A + idx;

	for (int j = 0; j < n; j++){
	  double temp = 0;
	  for (int i = 0;i < k; i++){
	    double a = *(A + i * lda);
	    double b = *(B + j * ldb + i);
	    temp = temp + a * b;
	  }
	  *(C + j * ldc + idx) = temp;
	}
}

__global__ void
dgemm_kernel2_1(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                        
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;
  double a = 0;
  double b1 = 0;
  double b2 = 0;
  for (int i = 0; i < k; i++){
    a = *(A + i * lda);
    b1 = *B;
    b2 = *(B + ldb);

    temp1 = temp1 + a * *(B + i);
    temp2 = temp2 + a * *(B + i + ldb);
  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
  
}

__global__ void
dgemm_kernel3(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;
  double a = 0;

  for (int j = 0; j < k; j += T){
    cache[threadIdx.x * 2] = *(B + threadIdx.x);
    cache[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;
    for (int i = 0; i < T; i++) {
      a = *(A + (i + j) * lda);
      temp1 += a * cache[i * 2];
      temp2 += a * cache[i * 2 + 1];
    }
    __syncthreads();

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;

}

__global__ void
dgemm_kernel4(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  double * cacheA = cache;
  double * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;

//prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  double r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;

    if (j + T < k) {  
      A = A + T * lda;
      
      r0 = *(A + 0 *lda);
      r1 = *(A + 1 *lda);
      r2 = *(A + 2 *lda);
      r3 = *(A + 3 *lda);   
      r4 = *(A + 4 *lda);
      r5 = *(A+ 5 *lda);
      r6 = *(A + 6 *lda);
      r7 = *(A + 7 *lda);

      r8 = *(A + 8 *lda);
      r9 = *(A + 9 *lda);
      r10 = *(A + 10 *lda);
      r11 = *(A + 11 *lda);
      r12 = *(A + 12 *lda);
      r13 = *(A + 13 *lda);
      r14 = *(A + 14 *lda);
      r15 = *(A + 15 *lda);
    }

    for (int i = 0; i < T; i++) {      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i * 2];
      temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 2 + 1];
    }
    if (j + T < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;

      cacheA[threadIdx.x + 8 * T] = r8;
      cacheA[threadIdx.x + 9 * T] = r9;
      cacheA[threadIdx.x + 10 * T] = r10;
      cacheA[threadIdx.x + 11 * T] = r11;
      cacheA[threadIdx.x + 12 * T] = r12;
      cacheA[threadIdx.x + 13 * T] = r13;
      cacheA[threadIdx.x + 14 * T] = r14;
      cacheA[threadIdx.x + 15 * T] = r15;
    }

  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;

}


//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, i, j, l (12)
//Double registers: cache, cacheA, cacheB, A, B, C, r0-3, temp1-2 (22)
//Shared mem.: T*2 + T*T (double)
__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ double cache[];
 
  double * cacheA = cache;
  double * cacheB = cache + T * t; //32 threads * 8 elements

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;

  //prefectch A 
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  A += t * lda;

  double r0, r1, r2, r3;//,r4,r5,r6,r7;

  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;


    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        r0 = *(A + 0 *lda);
        r1 = *(A + 1 *lda);
        r2 = *(A + 2 *lda);
        r3 = *(A + 3 *lda); 
      }

      #pragma unroll
      for (int i = 0; i < t; i++) {
      	temp1 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i ];
      	temp2 += cacheA[threadIdx.x +i * T] * cacheB[l - j + i + 1];
      }
      if (l + t < k) {
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      }
      A += t * lda;
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
    
}


//Single registers: m, n, k, T, t, lda, ldb, ldc, idx, i, j, l (12)
//Double registers: cache, cacheA, cacheB, A, B, C, nr0-3, cr0-3, temp1-2 (30)
//Shared mem.: T*2 + T*T (double)
__global__ void
dgemm_kernel4_2(int m, int n, int k, int T, int t, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ double cacheB[];

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  double temp2 = 0;

  double nr0, nr1, nr2, nr3;
  double cr0, cr1, cr2, cr3;

  //prefectch A 
  cr0 = *(A + 0 * lda);
  cr1 = *(A + 1 * lda);
  cr2 = *(A + 2 * lda);
  cr3 = *(A + 3 * lda);
  A += t * lda;

  for (int j = 0; j < k; j += T){ 
    __syncthreads();
    cacheB[threadIdx.x * 2] = *(B + threadIdx.x);
    cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    B += T;


    for (int l = j; l < j + T; l += t){
      if (l + t < k) {
        nr0 = *(A + 0 *lda);
        nr1 = *(A + 1 *lda);
        nr2 = *(A + 2 *lda);
        nr3 = *(A + 3 *lda); 
      }

      temp1 += cr0 * cacheB[l - j + 0 ];
      temp2 += cr0 * cacheB[l - j + 0 + 1];

      temp1 += cr1 * cacheB[l - j + 1 ];
      temp2 += cr1 * cacheB[l - j + 1 + 1];

      temp1 += cr2 * cacheB[l - j + 2 ];
      temp2 += cr2 * cacheB[l - j + 2 + 1];

      temp1 += cr3 * cacheB[l - j + 3 ];
      temp2 += cr3 * cacheB[l - j + 3 + 1];

      if (l + t < k) {
        cr0 = nr0;
        cr1 = nr1;
        cr2 = nr2;
        cr3 = nr3;
      }
      A += t * lda;
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  *(C + 1 * ldc + idx) = temp2;
    
}
