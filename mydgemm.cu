#include "hip/hip_runtime.h"
/*
    Enhanced Online ABFT
    UC Riverside
    Jieyang Chen
*/
#include "FT.h"
#include "common_magma.h"
#include "magma.h"
#include <stdlib.h>


#define NB 512
// encoding checksum for A
#define B 16
#define rB 8
#define cB 64
#define N 30720

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	/* logrithm reduction */
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}


__global__ void
chkenc_kernel1_5(double * A, int lda, double * Chk , int ldchk)
{

	//blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();


	double sum = 0;
	if (threadIdx.x == 0) {

		for (int i = 0; i < NB; i++) {
			sum += cache[i];
		}
		*(Chk + blockIdx.x * ldchk) = sum;
	}

	__syncthreads();

	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();


	sum = 0;
	if (threadIdx.x == 0) {

		for (int i = 0; i < NB; i++) {
			sum += cache[i];
		}
		*(Chk + blockIdx.x * ldchk + 1) = sum;
	}
	
}

__global__ void
chkenc_kernel2(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * NB + threadIdx.x;

	A = A + idx * lda;

	double temp = 0;
	double temp2 = 0;
	for (int i = 0; i < NB; i++) {
		temp += A[i];
		temp2 += A[i] * (i+1);
	}
	*(Chk + idx * ldchk) = temp;
	*(Chk + idx * ldchk+1) = temp2;
	
}

//N=32
__global__ void
chkenc_kernel3(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * B;

    double sum1 = 0;
    double sum2 = 0;

    double temp = 0;

	A = A + idx * lda;



	__shared__ double cache[B][B];

	for (int i = 0; i < NB; i += B) {
		
		//load a block to cache
		
			for (int j = 0; j < B; j++) {
				cache[threadIdx.x][j] = *(A + j * lda + threadIdx.x);
			}

		__syncthreads();

		for (int j = 0; j < B; j++) {
			temp = cache[j][threadIdx.x];
			sum1 += temp;
			sum2 += temp * (i + j + 1);
			
		}
		
		__syncthreads();

		A = A + B;
	}

	idx += threadIdx.x;

	*(Chk + idx * ldchk) = sum1;
	*(Chk + idx * ldchk+1) = sum2;
	
}


//N=16
__global__ void
chkenc_kernel3_P(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * B;

    double sum1 = 0;
    double sum2 = 0;

    double temp = 0;

	A = A + idx * lda;

	

	__shared__ double cache[B][B];


	double r0 = *(A + 0 * lda + threadIdx.x);
	double r1 = *(A + 1 * lda + threadIdx.x);
	double r2 = *(A + 2 * lda + threadIdx.x);
	double r3 = *(A + 3 * lda + threadIdx.x);
	double r4 = *(A + 4 * lda + threadIdx.x);
	double r5 = *(A + 5 * lda + threadIdx.x);
	double r6 = *(A + 6 * lda + threadIdx.x);
	double r7 = *(A + 7 * lda + threadIdx.x);
	
	double r8 = *(A + 8 * lda + threadIdx.x);
	double r9 = *(A + 9 * lda + threadIdx.x);
	double r10 = *(A + 10 * lda + threadIdx.x);
	double r11 = *(A + 11 * lda + threadIdx.x);
	double r12 = *(A + 12 * lda + threadIdx.x);
	double r13 = *(A + 13 * lda + threadIdx.x);
	double r14 = *(A + 14 * lda + threadIdx.x);
	double r15 = *(A + 15 * lda + threadIdx.x);
	/*
	double r16 = *(A + 16 * lda + threadIdx.x);
	double r17 = *(A + 17 * lda + threadIdx.x);
	double r18 = *(A + 18 * lda + threadIdx.x);
	double r19 = *(A + 19 * lda + threadIdx.x);
	double r20 = *(A + 20 * lda + threadIdx.x);
	double r21 = *(A + 21 * lda + threadIdx.x);
	double r22 = *(A + 22 * lda + threadIdx.x);
	double r23 = *(A + 23 * lda + threadIdx.x);
	double r24 = *(A + 24 * lda + threadIdx.x);
	double r25 = *(A + 25 * lda + threadIdx.x);
	double r26 = *(A + 26 * lda + threadIdx.x);
	double r27 = *(A + 27 * lda + threadIdx.x);
	double r28 = *(A + 28 * lda + threadIdx.x);
	double r29 = *(A + 29 * lda + threadIdx.x);
	double r30 = *(A + 30 * lda + threadIdx.x);
	double r31 = *(A + 31 * lda + threadIdx.x);
	*/



	for (int i = 0; i < NB; i += B) {

		//load current register->shared mem.
		cache[threadIdx.x][0] = r0;
		cache[threadIdx.x][1] = r1;
		cache[threadIdx.x][2] = r2;
		cache[threadIdx.x][3] = r3;
		cache[threadIdx.x][4] = r4;
		cache[threadIdx.x][5] = r5;
		cache[threadIdx.x][6] = r6;
		cache[threadIdx.x][7] = r7;
		
		cache[threadIdx.x][8] = r8;
		cache[threadIdx.x][9] = r9;
		cache[threadIdx.x][10] = r10;
		cache[threadIdx.x][11] = r11;
		cache[threadIdx.x][12] = r12;
		cache[threadIdx.x][13] = r13;
		cache[threadIdx.x][14] = r14;
		cache[threadIdx.x][15] = r15;
		/*
		cache[threadIdx.x][16] = r16;
		cache[threadIdx.x][17] = r17;
		cache[threadIdx.x][18] = r18;
		cache[threadIdx.x][19] = r19;
		cache[threadIdx.x][20] = r20;
		cache[threadIdx.x][21] = r21;
		cache[threadIdx.x][22] = r22;
		cache[threadIdx.x][23] = r23;
		cache[threadIdx.x][24] = r24;
		cache[threadIdx.x][25] = r25;
		cache[threadIdx.x][26] = r26;
		cache[threadIdx.x][27] = r27;
		cache[threadIdx.x][28] = r28;
		cache[threadIdx.x][29] = r29;
		cache[threadIdx.x][30] = r30;
		cache[threadIdx.x][31] = r31;
		*/

		__syncthreads();

		A = A + B;

		//load a next block to register
		
		 r0 = *(A + 0 * lda + threadIdx.x);
		 r1 = *(A + 1 * lda + threadIdx.x);
		 r2 = *(A + 2 * lda + threadIdx.x);
		 r3 = *(A + 3 * lda + threadIdx.x);
		 r4 = *(A + 4 * lda + threadIdx.x);
		 r5 = *(A + 5 * lda + threadIdx.x);
		 r6 = *(A + 6 * lda + threadIdx.x);
		 r7 = *(A + 7 * lda + threadIdx.x);
		 
		 r8 = *(A + 8 * lda + threadIdx.x);
		 r9 = *(A + 9 * lda + threadIdx.x);
		 r10 = *(A + 10 * lda + threadIdx.x);
		 r11 = *(A + 11 * lda + threadIdx.x);
		 r12 = *(A + 12 * lda + threadIdx.x);
		 r13 = *(A + 13 * lda + threadIdx.x);
		 r14 = *(A + 14 * lda + threadIdx.x);
		 r15 = *(A + 15 * lda + threadIdx.x);
		 /*
		 r16 = *(A + 16 * lda + threadIdx.x);
		 r17 = *(A + 17 * lda + threadIdx.x);
		 r18 = *(A + 18 * lda + threadIdx.x);
		 r19 = *(A + 19 * lda + threadIdx.x);
		 r20 = *(A + 20 * lda + threadIdx.x);
		 r21 = *(A + 21 * lda + threadIdx.x);
		 r22 = *(A + 22 * lda + threadIdx.x);
		 r23 = *(A + 23 * lda + threadIdx.x);
		 r24 = *(A + 24 * lda + threadIdx.x);
		 r25 = *(A + 25 * lda + threadIdx.x);
		 r26 = *(A + 26 * lda + threadIdx.x);
		 r27 = *(A + 27 * lda + threadIdx.x);
		 r28 = *(A + 28 * lda + threadIdx.x);
		 r29 = *(A + 29 * lda + threadIdx.x);
		 r30 = *(A + 30 * lda + threadIdx.x);
		 r31 = *(A + 31 * lda + threadIdx.x);
		 */


		for (int j = 0; j < B; j++) {
			temp = cache[j][threadIdx.x];
			sum1 += temp;
			sum2 += temp * (i + j + 1);
			
		}
		
		__syncthreads();

		
	}

	idx += threadIdx.x;

	*(Chk + idx * ldchk) = sum1;
	*(Chk + idx * ldchk+1) = sum2;
	
}


__global__ void
chkenc_kernel3_P_R(double * A, int lda, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    int idx = blockIdx.x * B;

    double sum1 = 0;
    double sum2 = 0;

    double temp = 0;

	A = A + idx;

	

	__shared__ double cache[B][B];


	double r0 = *(A + 0 * lda + threadIdx.x);
	double r1 = *(A + 1 * lda + threadIdx.x);
	double r2 = *(A + 2 * lda + threadIdx.x);
	double r3 = *(A + 3 * lda + threadIdx.x);
	double r4 = *(A + 4 * lda + threadIdx.x);
	double r5 = *(A + 5 * lda + threadIdx.x);
	double r6 = *(A + 6 * lda + threadIdx.x);
	double r7 = *(A + 7 * lda + threadIdx.x);
	
	double r8 = *(A + 8 * lda + threadIdx.x);
	double r9 = *(A + 9 * lda + threadIdx.x);
	double r10 = *(A + 10 * lda + threadIdx.x);
	double r11 = *(A + 11 * lda + threadIdx.x);
	double r12 = *(A + 12 * lda + threadIdx.x);
	double r13 = *(A + 13 * lda + threadIdx.x);
	double r14 = *(A + 14 * lda + threadIdx.x);
	double r15 = *(A + 15 * lda + threadIdx.x);
	/*
	double r16 = *(A + 16 * lda + threadIdx.x);
	double r17 = *(A + 17 * lda + threadIdx.x);
	double r18 = *(A + 18 * lda + threadIdx.x);
	double r19 = *(A + 19 * lda + threadIdx.x);
	double r20 = *(A + 20 * lda + threadIdx.x);
	double r21 = *(A + 21 * lda + threadIdx.x);
	double r22 = *(A + 22 * lda + threadIdx.x);
	double r23 = *(A + 23 * lda + threadIdx.x);
	double r24 = *(A + 24 * lda + threadIdx.x);
	double r25 = *(A + 25 * lda + threadIdx.x);
	double r26 = *(A + 26 * lda + threadIdx.x);
	double r27 = *(A + 27 * lda + threadIdx.x);
	double r28 = *(A + 28 * lda + threadIdx.x);
	double r29 = *(A + 29 * lda + threadIdx.x);
	double r30 = *(A + 30 * lda + threadIdx.x);
	double r31 = *(A + 31 * lda + threadIdx.x);
	*/



	for (int i = 0; i < NB; i += B) {

		//load current register->shared mem.
		cache[0][threadIdx.x] = r0;
		cache[1][threadIdx.x] = r1;
		cache[2][threadIdx.x] = r2;
		cache[3][threadIdx.x] = r3;
		cache[4][threadIdx.x] = r4;
		cache[5][threadIdx.x] = r5;
		cache[6][threadIdx.x] = r6;
		cache[7][threadIdx.x] = r7;
		
		cache[8][threadIdx.x] = r8;
		cache[9][threadIdx.x] = r9;
		cache[10][threadIdx.x] = r10;
		cache[11][threadIdx.x] = r11;
		cache[12][threadIdx.x] = r12;
		cache[13][threadIdx.x] = r13;
		cache[14][threadIdx.x] = r14;
		cache[15][threadIdx.x] = r15;
		/*
		cache[threadIdx.x][16] = r16;
		cache[threadIdx.x][17] = r17;
		cache[threadIdx.x][18] = r18;
		cache[threadIdx.x][19] = r19;
		cache[threadIdx.x][20] = r20;
		cache[threadIdx.x][21] = r21;
		cache[threadIdx.x][22] = r22;
		cache[threadIdx.x][23] = r23;
		cache[threadIdx.x][24] = r24;
		cache[threadIdx.x][25] = r25;
		cache[threadIdx.x][26] = r26;
		cache[threadIdx.x][27] = r27;
		cache[threadIdx.x][28] = r28;
		cache[threadIdx.x][29] = r29;
		cache[threadIdx.x][30] = r30;
		cache[threadIdx.x][31] = r31;
		*/

		__syncthreads();

		A = A + B * lda;

		//load a next block to register
		
		 r0 = *(A + 0 * lda + threadIdx.x);
		 r1 = *(A + 1 * lda + threadIdx.x);
		 r2 = *(A + 2 * lda + threadIdx.x);
		 r3 = *(A + 3 * lda + threadIdx.x);
		 r4 = *(A + 4 * lda + threadIdx.x);
		 r5 = *(A + 5 * lda + threadIdx.x);
		 r6 = *(A + 6 * lda + threadIdx.x);
		 r7 = *(A + 7 * lda + threadIdx.x);
		 
		 r8 = *(A + 8 * lda + threadIdx.x);
		 r9 = *(A + 9 * lda + threadIdx.x);
		 r10 = *(A + 10 * lda + threadIdx.x);
		 r11 = *(A + 11 * lda + threadIdx.x);
		 r12 = *(A + 12 * lda + threadIdx.x);
		 r13 = *(A + 13 * lda + threadIdx.x);
		 r14 = *(A + 14 * lda + threadIdx.x);
		 r15 = *(A + 15 * lda + threadIdx.x);
		 /*
		 r16 = *(A + 16 * lda + threadIdx.x);
		 r17 = *(A + 17 * lda + threadIdx.x);
		 r18 = *(A + 18 * lda + threadIdx.x);
		 r19 = *(A + 19 * lda + threadIdx.x);
		 r20 = *(A + 20 * lda + threadIdx.x);
		 r21 = *(A + 21 * lda + threadIdx.x);
		 r22 = *(A + 22 * lda + threadIdx.x);
		 r23 = *(A + 23 * lda + threadIdx.x);
		 r24 = *(A + 24 * lda + threadIdx.x);
		 r25 = *(A + 25 * lda + threadIdx.x);
		 r26 = *(A + 26 * lda + threadIdx.x);
		 r27 = *(A + 27 * lda + threadIdx.x);
		 r28 = *(A + 28 * lda + threadIdx.x);
		 r29 = *(A + 29 * lda + threadIdx.x);
		 r30 = *(A + 30 * lda + threadIdx.x);
		 r31 = *(A + 31 * lda + threadIdx.x);
		 */


		for (int j = 0; j < B; j++) {
			temp = cache[j][threadIdx.x];
			sum1 += temp;
			sum2 += temp * (i + j + 1);
			
		}
		
		__syncthreads();

		
	}

	idx += threadIdx.x;

	*(Chk + idx) = sum1;
	*(Chk + idx + ldchk) = sum2;
	
}


//N=16 Prefetch - full 
__global__ void
chkenc_kernel3_P_F(double * A, int lda, int nb, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    //int idx = blockIdx.x;


    double sum1 = 0;
    double sum2 = 0;

    double temp = 0;

	A = A + blockIdx.x * nb + blockIdx.y * nb * lda;

	

	__shared__ double cache[B][B];

	double r0 = 0;
	double r1 = 0;
	
	double r2 = 0;
	double r3 = 0;
	double r4 = 0;
	double r5 = 0;
	double r6 = 0;
	double r7 = 0;
	
	double r8 = 0;
	double r9 = 0;
	double r10 = 0;
	double r11 = 0;
	double r12 = 0;
	double r13 = 0;
	double r14 = 0;
	double r15 = 0;
	/*
	double r16 = 0;
	double r17 = 0;
	double r18 = 0;
	double r19 = 0;
	double r20 = 0;
	double r21 = 0;
	double r22 = 0;
	double r23 = 0;
	double r24 = 0;
	double r25 = 0;
	double r26 = 0;
	double r27 = 0;
	double r28 = 0;
	double r29 = 0;
	double r30 = 0;
	double r31 = 0;
	*/

	double * tA = A;
	for (int k = 0; k < nb; k += B) {
		
		r0 = *(A + 0 * lda + threadIdx.x);
		r1 = *(A + 1 * lda + threadIdx.x);
		
		r2 = *(A + 2 * lda + threadIdx.x);
		r3 = *(A + 3 * lda + threadIdx.x);
		r4 = *(A + 4 * lda + threadIdx.x);
		r5 = *(A + 5 * lda + threadIdx.x);
		r6 = *(A + 6 * lda + threadIdx.x);
		r7 = *(A + 7 * lda + threadIdx.x);
		
		r8 = *(A + 8 * lda + threadIdx.x);
		r9 = *(A + 9 * lda + threadIdx.x);
		r10 = *(A + 10 * lda + threadIdx.x);
		r11 = *(A + 11 * lda + threadIdx.x);
		r12 = *(A + 12 * lda + threadIdx.x);
		r13 = *(A + 13 * lda + threadIdx.x);
		r14 = *(A + 14 * lda + threadIdx.x);
		r15 = *(A + 15 * lda + threadIdx.x);
		/*
		r16 = *(A + 16 * lda + threadIdx.x);
		r17 = *(A + 17 * lda + threadIdx.x);
		r18 = *(A + 18 * lda + threadIdx.x);
		r19 = *(A + 19 * lda + threadIdx.x);
		r20 = *(A + 20 * lda + threadIdx.x);
		r21 = *(A + 21 * lda + threadIdx.x);
		r22 = *(A + 22 * lda + threadIdx.x);
		r23 = *(A + 23 * lda + threadIdx.x);
		r24 = *(A + 24 * lda + threadIdx.x);
		r25 = *(A + 25 * lda + threadIdx.x);
		r26 = *(A + 26 * lda + threadIdx.x);
		r27 = *(A + 27 * lda + threadIdx.x);
		r28 = *(A + 28 * lda + threadIdx.x);
		r29 = *(A + 29 * lda + threadIdx.x);
		r30 = *(A + 30 * lda + threadIdx.x);
		r31 = *(A + 31 * lda + threadIdx.x);
		*/

		sum1 = 0;
		sum2 = 0;
		temp = 0;


		for (int i = 0; i < nb; i += B) {

			//load current register->shared mem.
			cache[threadIdx.x][0] = r0;
			cache[threadIdx.x][1] = r1;
			
			cache[threadIdx.x][2] = r2;
			cache[threadIdx.x][3] = r3;
			cache[threadIdx.x][4] = r4;
			cache[threadIdx.x][5] = r5;
			cache[threadIdx.x][6] = r6;
			cache[threadIdx.x][7] = r7;
			
			cache[threadIdx.x][8] = r8;
			cache[threadIdx.x][9] = r9;
			cache[threadIdx.x][10] = r10;
			cache[threadIdx.x][11] = r11;
			cache[threadIdx.x][12] = r12;
			cache[threadIdx.x][13] = r13;
			cache[threadIdx.x][14] = r14;
			cache[threadIdx.x][15] = r15;
			/*
			cache[threadIdx.x][16] = r16;
			cache[threadIdx.x][17] = r17;
			cache[threadIdx.x][18] = r18;
			cache[threadIdx.x][19] = r19;
			cache[threadIdx.x][20] = r20;
			cache[threadIdx.x][21] = r21;
			cache[threadIdx.x][22] = r22;
			cache[threadIdx.x][23] = r23;
			cache[threadIdx.x][24] = r24;
			cache[threadIdx.x][25] = r25;
			cache[threadIdx.x][26] = r26;
			cache[threadIdx.x][27] = r27;
			cache[threadIdx.x][28] = r28;
			cache[threadIdx.x][29] = r29;
			cache[threadIdx.x][30] = r30;
			cache[threadIdx.x][31] = r31;
			*/

			__syncthreads();

			A = A + B;

			//load a next block to register
			
			 r0 = *(A + 0 * lda + threadIdx.x);
			 r1 = *(A + 1 * lda + threadIdx.x);
			 
			 r2 = *(A + 2 * lda + threadIdx.x);
			 r3 = *(A + 3 * lda + threadIdx.x);
			 r4 = *(A + 4 * lda + threadIdx.x);
			 r5 = *(A + 5 * lda + threadIdx.x);
			 r6 = *(A + 6 * lda + threadIdx.x);
			 r7 = *(A + 7 * lda + threadIdx.x);
			 
			 r8 = *(A + 8 * lda + threadIdx.x);
			 r9 = *(A + 9 * lda + threadIdx.x);
			 r10 = *(A + 10 * lda + threadIdx.x);
			 r11 = *(A + 11 * lda + threadIdx.x);
			 r12 = *(A + 12 * lda + threadIdx.x);
			 r13 = *(A + 13 * lda + threadIdx.x);
			 r14 = *(A + 14 * lda + threadIdx.x);
			 r15 = *(A + 15 * lda + threadIdx.x);
			/*
			 r16 = *(A + 16 * lda + threadIdx.x);
			 r17 = *(A + 17 * lda + threadIdx.x);
			 r18 = *(A + 18 * lda + threadIdx.x);
			 r19 = *(A + 19 * lda + threadIdx.x);
			 r20 = *(A + 20 * lda + threadIdx.x);
			 r21 = *(A + 21 * lda + threadIdx.x);
			 r22 = *(A + 22 * lda + threadIdx.x);
			 r23 = *(A + 23 * lda + threadIdx.x);
			 r24 = *(A + 24 * lda + threadIdx.x);
			 r25 = *(A + 25 * lda + threadIdx.x);
			 r26 = *(A + 26 * lda + threadIdx.x);
			 r27 = *(A + 27 * lda + threadIdx.x);
			 r28 = *(A + 28 * lda + threadIdx.x);
			 r29 = *(A + 29 * lda + threadIdx.x);
			 r30 = *(A + 30 * lda + threadIdx.x);
			 r31 = *(A + 31 * lda + threadIdx.x);
			 */


			for (int j = 0; j < B; j++) {
				temp = cache[j][threadIdx.x];
				sum1 += temp;
				sum2 += temp * (i + j + 1);
				
			}
			
			__syncthreads();

			
		}

		//idx += threadIdx.x;

		*(Chk + (blockIdx.y * nb + k + threadIdx.x) * ldchk + blockIdx.x * 2 ) = sum1;
		*(Chk + (blockIdx.y * nb + k + threadIdx.x) * ldchk + blockIdx.x * 2 + 1) = sum2;


		tA += B * lda;
		//if(threadIdx.x == 0)
		//printf("next:%f\n", (*tA));
		A = tA ;
	}
	
}


//N=16 Prefetch - full - Row
__global__ void
chkenc_kernel3_P_FR(double * A, int lda, int nb, double * Chk , int ldchk)
{

    //blockIdx.x: determin the column to process
    //int idx = blockIdx.x;


    double sum1 = 0;
    double sum2 = 0;

    double temp = 0;

	A = A + blockIdx.x * nb + blockIdx.y * nb * lda;

	

	__shared__ double cache[B][B];

	double r0 = 0;
	double r1 = 0;
	
	double r2 = 0;
	double r3 = 0;
	double r4 = 0;
	double r5 = 0;
	double r6 = 0;
	double r7 = 0;
	
	double r8 = 0;
	double r9 = 0;
	double r10 = 0;
	double r11 = 0;
	double r12 = 0;
	double r13 = 0;
	double r14 = 0;
	double r15 = 0;
	/*
	double r16 = 0;
	double r17 = 0;
	double r18 = 0;
	double r19 = 0;
	double r20 = 0;
	double r21 = 0;
	double r22 = 0;
	double r23 = 0;
	double r24 = 0;
	double r25 = 0;
	double r26 = 0;
	double r27 = 0;
	double r28 = 0;
	double r29 = 0;
	double r30 = 0;
	double r31 = 0;
	*/

	double * tA = A;
	for (int k = 0; k < nb; k += B) {
		
		r0 = *(A + 0 * lda + threadIdx.x);
		r1 = *(A + 1 * lda + threadIdx.x);
		
		r2 = *(A + 2 * lda + threadIdx.x);
		r3 = *(A + 3 * lda + threadIdx.x);
		r4 = *(A + 4 * lda + threadIdx.x);
		r5 = *(A + 5 * lda + threadIdx.x);
		r6 = *(A + 6 * lda + threadIdx.x);
		r7 = *(A + 7 * lda + threadIdx.x);
		
		r8 = *(A + 8 * lda + threadIdx.x);
		r9 = *(A + 9 * lda + threadIdx.x);
		r10 = *(A + 10 * lda + threadIdx.x);
		r11 = *(A + 11 * lda + threadIdx.x);
		r12 = *(A + 12 * lda + threadIdx.x);
		r13 = *(A + 13 * lda + threadIdx.x);
		r14 = *(A + 14 * lda + threadIdx.x);
		r15 = *(A + 15 * lda + threadIdx.x);
		/*
		r16 = *(A + 16 * lda + threadIdx.x);
		r17 = *(A + 17 * lda + threadIdx.x);
		r18 = *(A + 18 * lda + threadIdx.x);
		r19 = *(A + 19 * lda + threadIdx.x);
		r20 = *(A + 20 * lda + threadIdx.x);
		r21 = *(A + 21 * lda + threadIdx.x);
		r22 = *(A + 22 * lda + threadIdx.x);
		r23 = *(A + 23 * lda + threadIdx.x);
		r24 = *(A + 24 * lda + threadIdx.x);
		r25 = *(A + 25 * lda + threadIdx.x);
		r26 = *(A + 26 * lda + threadIdx.x);
		r27 = *(A + 27 * lda + threadIdx.x);
		r28 = *(A + 28 * lda + threadIdx.x);
		r29 = *(A + 29 * lda + threadIdx.x);
		r30 = *(A + 30 * lda + threadIdx.x);
		r31 = *(A + 31 * lda + threadIdx.x);
		*/

		sum1 = 0;
		sum2 = 0;
		temp = 0;


		for (int i = 0; i < nb; i += B) {

			//load current register->shared mem.
			cache[0][threadIdx.x] = r0;
			cache[1][threadIdx.x] = r1;
			
			cache[2][threadIdx.x] = r2;
			cache[3][threadIdx.x] = r3;
			cache[4][threadIdx.x] = r4;
			cache[5][threadIdx.x] = r5;
			cache[6][threadIdx.x] = r6;
			cache[7][threadIdx.x] = r7;
			
			cache[8][threadIdx.x] = r8;
			cache[9][threadIdx.x] = r9;
			cache[10][threadIdx.x] = r10;
			cache[11][threadIdx.x] = r11;
			cache[12][threadIdx.x] = r12;
			cache[13][threadIdx.x] = r13;
			cache[14][threadIdx.x] = r14;
			cache[15][threadIdx.x] = r15;
			/*
			cache[threadIdx.x][16] = r16;
			cache[threadIdx.x][17] = r17;
			cache[threadIdx.x][18] = r18;
			cache[threadIdx.x][19] = r19;
			cache[threadIdx.x][20] = r20;
			cache[threadIdx.x][21] = r21;
			cache[threadIdx.x][22] = r22;
			cache[threadIdx.x][23] = r23;
			cache[threadIdx.x][24] = r24;
			cache[threadIdx.x][25] = r25;
			cache[threadIdx.x][26] = r26;
			cache[threadIdx.x][27] = r27;
			cache[threadIdx.x][28] = r28;
			cache[threadIdx.x][29] = r29;
			cache[threadIdx.x][30] = r30;
			cache[threadIdx.x][31] = r31;
			*/

			__syncthreads();

			A = A + B * lda;

			//load a next block to register
			
			 r0 = *(A + 0 * lda + threadIdx.x);
			 r1 = *(A + 1 * lda + threadIdx.x);
			 
			 r2 = *(A + 2 * lda + threadIdx.x);
			 r3 = *(A + 3 * lda + threadIdx.x);
			 r4 = *(A + 4 * lda + threadIdx.x);
			 r5 = *(A + 5 * lda + threadIdx.x);
			 r6 = *(A + 6 * lda + threadIdx.x);
			 r7 = *(A + 7 * lda + threadIdx.x);
			 
			 r8 = *(A + 8 * lda + threadIdx.x);
			 r9 = *(A + 9 * lda + threadIdx.x);
			 r10 = *(A + 10 * lda + threadIdx.x);
			 r11 = *(A + 11 * lda + threadIdx.x);
			 r12 = *(A + 12 * lda + threadIdx.x);
			 r13 = *(A + 13 * lda + threadIdx.x);
			 r14 = *(A + 14 * lda + threadIdx.x);
			 r15 = *(A + 15 * lda + threadIdx.x);
			/*
			 r16 = *(A + 16 * lda + threadIdx.x);
			 r17 = *(A + 17 * lda + threadIdx.x);
			 r18 = *(A + 18 * lda + threadIdx.x);
			 r19 = *(A + 19 * lda + threadIdx.x);
			 r20 = *(A + 20 * lda + threadIdx.x);
			 r21 = *(A + 21 * lda + threadIdx.x);
			 r22 = *(A + 22 * lda + threadIdx.x);
			 r23 = *(A + 23 * lda + threadIdx.x);
			 r24 = *(A + 24 * lda + threadIdx.x);
			 r25 = *(A + 25 * lda + threadIdx.x);
			 r26 = *(A + 26 * lda + threadIdx.x);
			 r27 = *(A + 27 * lda + threadIdx.x);
			 r28 = *(A + 28 * lda + threadIdx.x);
			 r29 = *(A + 29 * lda + threadIdx.x);
			 r30 = *(A + 30 * lda + threadIdx.x);
			 r31 = *(A + 31 * lda + threadIdx.x);
			 */


			for (int j = 0; j < B; j++) {
				temp = cache[j][threadIdx.x];
				sum1 += temp;
				sum2 += temp * (i + j + 1);
				
			}
			
			__syncthreads();

			
		}

		//idx += threadIdx.x;

		*(Chk + (blockIdx.y * 2) * ldchk + blockIdx.x * nb + k + threadIdx.x) = sum1;
		*(Chk + (blockIdx.y * 2 + 1) * ldchk + blockIdx.x * nb + k + threadIdx.x) = sum2;


		tA += B;
		//if(threadIdx.x == 0)
		//printf("next:%f\n", (*tA));
		A = tA ;
	}
	
}


//N=16
__global__ void
chkenc_kernel3_5(double * A, int lda, double * Chk, int ldchk)
{

    //blockIdx.x: determin the column to process
    

    int idx = blockIdx.x * B;

    double sum1 = 0;
    double sum2 = 0;

	A = A + idx * lda;

	__shared__ double cache[B][B]; //B * B

	for (int i = 0; i < NB; i += B) {
		
		//load a block to cache
		cache[threadIdx.y][threadIdx.x] = *(A + threadIdx.y * lda + threadIdx.x);
		__syncthreads();

		int k = B / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.y][threadIdx.x] += cache[threadIdx.y][threadIdx.x + k];
			}
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum1 += cache[threadIdx.y][0];
		}


		cache[threadIdx.y][threadIdx.x] = *(A + threadIdx.y * lda + threadIdx.x) * (i + threadIdx.x + 1);
		__syncthreads();
		k = B / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.y][threadIdx.x] += cache[threadIdx.y][threadIdx.x + k];
			}
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum2 += cache[threadIdx.y][0];
		}
				
		A = A + B;
	}

	idx += threadIdx.y;

	if (threadIdx.x == 0) {
		*(Chk + idx * ldchk) = sum1;
		*(Chk + idx * ldchk+1) = sum2;
	}
	
}


//N=16
__global__ void
chkenc_kernel3_5_P(double * A, int lda, double * Chk, int ldchk)
{

    //blockIdx.x: determin the column to process
    

    int idx = blockIdx.x * B;

    double sum = 0;

	A = A + idx * lda;
	idx += threadIdx.y;

	__shared__ double cache[B][B]; //B * B

	double r = *(A + threadIdx.y * lda + threadIdx.x);


	for (int i = 0; i < NB; i += B) {
		
		//load a block register -> cache
		cache[threadIdx.y][threadIdx.x] = r;
		__syncthreads();

		//load next to register
		r = *(A + i + B + threadIdx.y * lda + threadIdx.x);

		int k = B / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.y][threadIdx.x] += cache[threadIdx.y][threadIdx.x + k];
			}
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum += cache[threadIdx.y][0];
		}

	}

	if (threadIdx.x == 0) {
		*(Chk + idx * ldchk) = sum;
	}

	sum = 0;
	r = *(A + threadIdx.y * lda + threadIdx.x);

	for (int i = 0; i < NB; i += B) {

		cache[threadIdx.y][threadIdx.x] = r * (i + threadIdx.x + 1);
		__syncthreads();

		r = *(A + i + B +  threadIdx.y * lda + threadIdx.x);
		int k = B / 2;
		while (k != 0) {
			if (threadIdx.x < k) {
				cache[threadIdx.y][threadIdx.x] += cache[threadIdx.y][threadIdx.x + k];
			}
			__syncthreads();
			k /= 2;
		}
		if (threadIdx.x == 0) {
			sum += cache[threadIdx.y][0];
		}
	}

	

	if (threadIdx.x == 0) {
		*(Chk + idx * ldchk+1) = sum;
	}
	
}



void col_chkenc(double * A, int lda, int m, int n, int nb, double * chk , int ldchk, magma_queue_t stream) {
  /*  int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = 32; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel4, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	//printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );
	*/
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	//int rb = B;
	//int cb = B;
	dim3 d(m/NB, n/NB, 1);
	
	//for (int i = 0; i < m; i+=NB) {
	//	chkenc_kernel3_P<<<n/B, B, 0, stream>>>(A + i, lda, chk + (i/NB)*2, ldchk);
	//}
	//chkenc_kernel3_P_F<<<d, B, 0, stream>>>(A, lda, chk, ldchk);
	//chkenc_kernel3_P_R<<<m/B, B, 0, stream>>>(A, lda, chk, ldchk);

	chkenc_kernel3_P_F<<<d, B, 0, stream>>>(A, lda, nb, chk, ldchk);
}


void row_chkenc(double * A, int lda, int m, int n, int nb, double * chk , int ldchk, magma_queue_t stream) {
  /*  int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = 32; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel4, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	//printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );
	*/
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	//int rb = B;
	//int cb = B;
	dim3 d(m/nb, n/nb, 1);
	
	//for (int i = 0; i < m; i+=NB) {
	//	chkenc_kernel3_P<<<n/B, B, 0, stream>>>(A + i, lda, chk + (i/NB)*2, ldchk);
	//}
	//chkenc_kernel3_P_F<<<d, B, 0, stream>>>(A, lda, chk, ldchk);
	//chkenc_kernel3_P_R<<<m/B, B, 0, stream>>>(A, lda, chk, ldchk);

	chkenc_kernel3_P_FR<<<d, B, 0, stream>>>(A, lda, nb, chk, ldchk);
}



