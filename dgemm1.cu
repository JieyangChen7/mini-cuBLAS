
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include <time.h>
#include <stdio.h>
#define TEST_RUN 10 
#define ESP 10e-10
using namespace std;


__global__ void
dgemm_kernel2(int m, int n, int k, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel2_1(int m, int n, int k, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

__global__ void
dgemm_kernel3(int m, int n, int k, int T, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel4(int m, int n, int k, int T, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

void check_C(double * dC, int m, double * checkC);

void test_cublas_mv(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_cublas_mm(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_kernel2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel2_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_kernel3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4_1(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test(int m, int k);

int main(){
	for (int i = 128; i <= 32768; i *= 2){
		//i = 20480;
		cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 1 << ")" << endl;
		test(i, i);
	}
}

void test(int m, int k){
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 1;
    //int k = 20480;
    double * A = new double[m * k];
    double * B = new double[n * k];
    double * C = new double[m * n];  
    double * checkC = new double[m * n];   

    for (int i = 0;i < m * k; i++){
    	A[i] = i;
    }

    //    for (int i = 0; i < m; i++){
    // for (int j = 0; j < k; j++){
    //	cout << *( A + i + j * m) << " ";
    // }
    // cout << endl;
    //}
    
    for (int i = 0; i < n * k; i++){
    	B[i] = 1;
    }
    
    double * dA;
    hipMalloc(&dA, m * k * sizeof(double));
    int lda = m;

    double * dB; 
    hipMalloc(&dB,  n * k * sizeof(double));
    int ldb = k;

    double * dC;
    hipMalloc(&dC, m * n * sizeof(double));
    int ldc = m;

    double * dcheckC;
    hipMalloc(&dcheckC, m * n * sizeof(double));

    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    
   

    test_cublas_mm(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dcheckC, ldc);
/*
	test_cublas_mv(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

    test_kernel2(m, n, k, 
				 dA, lda, 
				 dB, ldb, 
				 dC, ldc);
    
	test_kernel2_1(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

	test_kernel3(m, n, k, 
				 dA, lda, 
				 dB, ldb, 
				 dC, ldc);
*/
	test_kernel4(m, n, k, 
				 dA, lda, 
				 dB, ldb, 
				 dC, ldc);

/*
	test_kernel4_1(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

*/
   
    hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(checkC, dcheckC, m * n * sizeof(double), hipMemcpyDeviceToHost);

	check_C(C, m, checkC);    
    
    //for (int i = 0; i < m * n; i++){
    // cout<<C[i]<<" ";	
    //}
    
    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    

}


void test_cublas_mv(int m, int n, int k, 
				 double * dA, int lda, 
				 double * dB, int ldb, 
				 double * dC, int ldc){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);


    int incb = 1;
    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemv(handle, HIPBLAS_OP_N, m, k,
      			  &one, dA, lda, dB, incb, &zero, dC, incb);

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of culasdgemv:" << real_time <<" ms." << endl;

}


void test_cublas_mm(int m, int n, int k, 
				 double * dA, int lda, 
				 double * dB, int ldb, 
				 double * dC, int ldc){

    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

 
    int incb = 1;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
	  	  &one, dA, lda, dB, ldb, &zero, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of culasdgemm:" << real_time <<" ms." << endl;
}

void test_kernel2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc){


    int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();

    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, 
							dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of dgemm_kernel2: " << real_time << " ms." << endl;    

} 


void test_kernel2_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){
	


	int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock(); 
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2_1<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
    						  dA, lda, dB, ldb, dC, ldc);


    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;

    cout <<"Runing time of dgemm_kernel2_1: " << real_time << " ms." << endl;


}

void test_kernel3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc){

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel3<<<blocksPerGrid, threadsPerBlock,  T * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    hipDeviceSynchronize();
    t = clock() - t;

    float real_time = ((float)t)/CLOCKS_PER_SEC;
    cout <<"Runing time of dgemm_kernel3: " << real_time << " ms." << endl;	    
}


void test_kernel4(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++) {
      dgemm_kernel4<<<blocksPerGrid, threadsPerBlock, ((T) + (T * T)) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    }
    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;
    cout <<"Runing time of dgemm_kernel4: " << real_time << " ms." << endl;    
}

void test_kernel4_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){


    
    int T = 32;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    clock_t t = clock();
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_1<<<blocksPerGrid, threadsPerBlock, ((T) + (T * (T/4))) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    

    hipDeviceSynchronize();
    t = clock() - t;
    float real_time = ((float)t)/CLOCKS_PER_SEC;


    cout <<"Runing time of dgemm_kernel4: " << real_time << " ms." << endl;	  

}

void check_C(double * dC, int m, double * checkC) {
	for (int i = 0; i < m; i++){
		if (abs(dC[i] - checkC[i]) < ESP){
			cout << "error:" << abs(dC[i] - checkC[i]) << endl;
			break;
		}
	}
	cout << "correct" << endl;
}


__global__ void
dgemm_kernel2(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
	//determine the row to process
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	A = A + idx;

	for (int j = 0; j < n; j++){
	  double temp = 0;
	  for (int i = 0;i < k; i++){
	    double a = *(A + i * lda);
	    double b = *(B + j * ldb + i);
	    temp = temp + a * b;
	  }
	  *(C + j * ldc + idx) = temp;
	}
}

__global__ void
dgemm_kernel2_1(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                        
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  double b1 = 0;
  //double b2 = 0;
  for (int i = 0; i < k; i++){
    A += lda;
    a = *A;

    B += 1;
    b1 = *B;
    //b2 = *(B + ldb);

    temp1 = temp1 + a * b1;
    //temp2 = temp2 + a * b2;
  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;
  
}

__global__ void
dgemm_kernel3(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  //double b1 = 0;
  //double b2 = 0;
  for (int j = 0; j < k; j += T){
    B += T;
    cache[threadIdx.x] = *(B + threadIdx.x);
    //cache[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    for (int i = 0; i < T; i++) {
      //i+j
      a = *(A + (i + j) * lda);
      //b1 = cache[i * 2]
      //b2 = cache[i * 2 + 1]
      temp1 += a * cache[i];
      //temp2 += a * cache[i * 2 + 1];
    }
    __syncthreads();

  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;

}

__global__ void
dgemm_kernel4(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  extern __shared__ double cache[];
  
  double * cacheA = cache;
  double * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
                                                                                                                                                                                      
  //prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  
  double r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11, r12, r13, r14, r15;

  for (int j = 0; j < k; j += T){
    __syncthreads();
    cacheB[threadIdx.x] = *(B + threadIdx.x);
    __syncthreads();
    B += T;
    
    A = A + T * lda;
    
    r0 = *(A + 0 *lda);
    r1 = *(A + 1 *lda);
    r2 = *(A + 2 *lda);
    r3 = *(A + 3 *lda);   
    r4 = *(A + 4 *lda);
    r5 = *(A+ 5 *lda);
    r6 = *(A + 6 *lda);
    r7 = *(A + 7 *lda);

    r8 = *(A + 8 *lda);
    r9 = *(A + 9 *lda);
    r10 = *(A + 10 *lda);
    r11 = *(A + 11 *lda);
    r12 = *(A + 12 *lda);
    r13 = *(A + 13 *lda);
    r14 = *(A + 14 *lda);
    r15 = *(A + 15 *lda);
 
    for (int i = 0; i < T; i++) {
      temp1 += cacheA[threadIdx.x + i * T] * cacheB[i];
    }

    cacheA[threadIdx.x + 0 * T] = r0;
    cacheA[threadIdx.x + 1 * T] = r1;
    cacheA[threadIdx.x + 2 * T] = r2;
    cacheA[threadIdx.x + 3 * T] = r3;
    cacheA[threadIdx.x + 4 * T] = r4;
    cacheA[threadIdx.x + 5 * T] = r5;
    cacheA[threadIdx.x + 6 * T] = r6;
    cacheA[threadIdx.x + 7 * T] = r7;

    cacheA[threadIdx.x + 8 * T] = r8;
    cacheA[threadIdx.x + 9 * T] = r9;
    cacheA[threadIdx.x + 10 * T] = r10;
    cacheA[threadIdx.x + 11 * T] = r11;
    cacheA[threadIdx.x + 12 * T] = r12;
    cacheA[threadIdx.x + 13 * T] = r13;
    cacheA[threadIdx.x + 14 * T] = r14;
    cacheA[threadIdx.x + 15 * T] = r15;

  }
  *(C + idx) = temp1;
}




__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ double cache[];
 
  double * cacheA = cache;
  double * cacheB = cache + T * (T / 4); //32 threads * 8 elements

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  //double b1 = 0；                                                                                                                                                                                                                                                                        
  //double b2 = 0;                                                                                                                                                                                                                                                                         

  //prefectch A 
  int t = T / 4;
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  //printf("%d: %f, %f\n", threadIdx.x, cacheA[threadIdx.x + 0 * T], cacheA[threadIdx.x + 1 * T]);
  __syncthreads();

  double r0, r1, r2,r3;//,r4,r5,r6,r7;
  double * orgB = B;
  double * orgA = A;
  for (int j = 0; j < k; j += T){ 
    B = orgB + j;
    __syncthreads();
    cacheB[threadIdx.x] = *(B + threadIdx.x);
    //cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    //printf("[iter=%d]%d: %f, %f\n", j, threadIdx.x, cacheB[threadIdx.x * 2], cacheB[threadIdx.x * 2 + 1]); 

    A = orgA + j * lda;
    for (int l = 0; l < 4; l++){
      
      r0 = *(A + (0 + (l+1) * t) *lda);
      r1 = *(A + (1 + (l+1) * t) *lda);
      r2 = *(A + (2 + (l+1) * t) *lda);
      r3 = *(A + (3 + (l+1) * t) *lda);
      /*r4 = *(A + (4 + (l+1) * t) *lda);
      r5 = *(A + (5 + (l+1) * t) *lda);
      r6 = *(A + (6 + (l+1) * t) *lda);
      r7 = *(A + (7 + (l+1) * t) *lda);
      */
      //__syncthreads();
      //printf("[iter=%d]%d: %f, %f\n", j+l*t, threadIdx.x, cacheA[threadIdx.x + 0 * T], cacheA[threadIdx.x + 1 * T]);
      for (int i = 0; i < t; i++) {
	       temp1 += cacheA[threadIdx.x +i * T] * cacheB[t * l + i ];
	       //temp2 += cacheA[threadIdx.x +i * T] * cacheB[t * l + i * 2 + 1];
      }
      
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      /*cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;
      */
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;
    
}
