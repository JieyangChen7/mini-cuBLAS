#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"
#include "papi.h"
#include <time.h>
#include <stdio.h>
#define TEST_RUN 10 
using namespace std;


__global__ void
dgemm_kernel2(int m, int n, int k, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel2_1(int m, int n, int k, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);

__global__ void
dgemm_kernel3(int m, int n, int k, int T, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel4(int m, int n, int k, int T, 
			  double * A, int lda, 
			  double * B, int ldb, 
			  double * C, int ldc);

__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, 
				double * A, int lda, 
				double * B, int ldb, 
				double * C, int ldc);


void test_cublas_mv(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_cublas_mm(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_kernel2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel2_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc);

void test_kernel3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test_kernel4_1(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc);

void test(int m, int k);

int main(){
	for (int i = 8192; i <= 32768; i *= 2){
		//i = 20480;
		cout << "Test on: A (" << i << " x " << i << ") by B (" << i << " x " << 1 << ")" << endl;
		test(i, i);
	}
}

void test(int m, int k){
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    //int m = 20480;
    int n = 1;
    //int k = 20480;
    double * A = new double[m * k];
    double * B = new double[n * k];
    double * C = new double[m * n];    

    for (int i = 0;i < m * k; i++){
    	A[i] = i;
    }

    //    for (int i = 0; i < m; i++){
    // for (int j = 0; j < k; j++){
    //	cout << *( A + i + j * m) << " ";
    // }
    // cout << endl;
    //}
    
    for (int i = 0; i < n * k; i++){
    	B[i] = 1;
    }
    
    double * dA;
    hipMalloc(&dA, m * k * sizeof(double));
    int lda = m;

    double * dB; 
    hipMalloc(&dB,  n * k * sizeof(double));
    int ldb = k;

    double * dC;
    hipMalloc(&dC, m * n * sizeof(double));
    int ldc = m;

    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    
   
/*
    test_cublas_mm(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

	test_cublas_mv(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

    test_kernel2(m, n, k, 
				 dA, lda, 
				 dB, ldb, 
				 dC, ldc);
    
	test_kernel2_1(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

	test_kernel3(m, n, k, 
				 dA, lda, 
				 dB, ldb, 
				 dC, ldc);
*/
	test_kernel4(m, n, k, 
				 dA, lda, 
				 dB, ldb, 
				 dC, ldc);

/*
	test_kernel4_1(m, n, k, 
				   dA, lda, 
				   dB, ldb, 
				   dC, ldc);

*/
   
    hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
    
    //for (int i = 0; i < m * n; i++){
    // cout<<C[i]<<" ";	
    //}
    
    //free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;
    

}


void test_cublas_mv(int m, int n, int k, 
				 double * dA, int lda, 
				 double * dB, int ldb, 
				 double * dC, int ldc){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEventRecord(start);
    int incb = 1;
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemv(handle, HIPBLAS_OP_N, m, k,
      			  &one, dA, lda, dB, incb, &zero, dC, incb);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);
    cout <<"Runing time of culasdgemv:" << real_time <<" ms." << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void test_cublas_mm(int m, int n, int k, 
				 double * dA, int lda, 
				 double * dB, int ldb, 
				 double * dC, int ldc){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


    double one = 1;
    double zero = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEventRecord(start);
    int incb = 1;
    for (int i = 0; i < TEST_RUN; i++)
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
	  	  &one, dA, lda, dB, ldb, &zero, dC, ldc);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);
    cout <<"Runing time of culasdgemm:" << real_time <<" ms." << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void test_kernel2(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, 
							dA, lda, dB, ldb, dC, ldc);
	hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);

    cout <<"Runing time of dgemm_kernel2: " << real_time << " ms." << endl;    
    hipEventDestroy(start);
    hipEventDestroy(stop);
} 


void test_kernel2_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int T = 128;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel2_1<<<blocksPerGrid, threadsPerBlock>>>(m, n, k,
    						  dA, lda, dB, ldb, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);

    cout <<"Runing time of dgemm_kernel2_1: " << real_time << " ms." << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void test_kernel3(int m, int n, int k, 
				  double * dA, int lda, 
				  double * dB, int ldb, 
				  double * dC, int ldc){

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;
    
    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel3<<<blocksPerGrid, threadsPerBlock,  T * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);

    cout <<"Runing time of dgemm_kernel3: " << real_time << " ms." << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);		    
}


void test_kernel4(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    int T = 16;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4<<<blocksPerGrid, threadsPerBlock, ((T) + (T * T)) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);

    cout <<"Runing time of dgemm_kernel4: " << real_time << " ms." << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);		    
}

void test_kernel4_1(int m, int n, int k, 
				    double * dA, int lda, 
				    double * dB, int ldb, 
				    double * dC, int ldc){

	
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    
    int T = 32;
    int blocksPerGrid = m / T;
    int threadsPerBlock = T;

    hipEventRecord(start);
    for (int i = 0; i < TEST_RUN; i++)
      dgemm_kernel4_1<<<blocksPerGrid, threadsPerBlock, ((T) + (T * (T/4))) * sizeof(double)>>>(m, n, k, T, dA, lda, dB, ldb, dC, ldc);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float real_time = 0;
    hipEventElapsedTime(&real_time, start, stop);

    cout <<"Runing time of dgemm_kernel4: " << real_time << " ms." << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);			  

}



__global__ void
dgemm_kernel2(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
	//determine the row to process
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	A = A + idx;

	for (int j = 0; j < n; j++){
	  double temp = 0;
	  for (int i = 0;i < k; i++){
	    double a = *(A + i * lda);
	    double b = *(B + j * ldb + i);
	    temp = temp + a * b;
	  }
	  *(C + j * ldc + idx) = temp;
	}
}

__global__ void
dgemm_kernel2_1(int m, int n, int k, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  //determine the row to process                                                        
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  double b1 = 0;
  //double b2 = 0;
  for (int i = 0; i < k; i++){
    A += lda;
    a = *A;

    B += 1;
    b1 = *B;
    //b2 = *(B + ldb);

    temp1 = temp1 + a * b1;
    //temp2 = temp2 + a * b2;
  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;
  
}

__global__ void
dgemm_kernel3(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  //double b1 = 0;
  //double b2 = 0;
  for (int j = 0; j < k; j += T){
    B += T;
    cache[threadIdx.x] = *(B + threadIdx.x);
    //cache[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    for (int i = 0; i < T; i++) {
      //i+j
      a = *(A + (i + j) * lda);
      //b1 = cache[i * 2]
      //b2 = cache[i * 2 + 1]
      temp1 += a * cache[i];
      //temp2 += a * cache[i * 2 + 1];
    }
    __syncthreads();

  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;

}

__global__ void
dgemm_kernel4(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)
  extern __shared__ double cache[];
  
  double * cacheA = cache;
  double * cacheB = cache + T * T;
  
  //determine the row to process
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  //double b1 = 0；      
  //double b2 = 0;                                                                                                                                                                                       
  
  //prefectch A
  for (int i = 0; i < T; i++){
    cacheA[threadIdx.x + i * T] = *(A + threadIdx.x + i * lda);
  }
  __syncthreads();
  
  double r0, r1, r2,r3,r4,r5,r6,r7,r8,r9,r10,r11,r12,r13,r14,r15;

  for (int j = 0; j < k; j += T){
    B += T;
    __syncthreads();
    cacheB[threadIdx.x] = *(B + threadIdx.x);
    //cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    
    A = A + T * lda;
    
    r0 = *(A + threadIdx.x + 0 *lda);
    r1 = *(A + threadIdx.x + 1 *lda);
    r2 = *(A + threadIdx.x + 2 *lda);
    r3 = *(A + threadIdx.x + 3 *lda);   
    r4 = *(A + threadIdx.x + 4 *lda);
    r5 = *(A + threadIdx.x + 5 *lda);
    r6 = *(A + threadIdx.x + 6 *lda);
    r7 = *(A + threadIdx.x + 7 *lda);

    r8 = *(A + threadIdx.x + 8 *lda);
    r9 = *(A + threadIdx.x + 9 *lda);
    r10 = *(A + threadIdx.x + 10 *lda);
    r11 = *(A + threadIdx.x + 11 *lda);
    r12 = *(A + threadIdx.x + 12 *lda);
    r13 = *(A + threadIdx.x + 13 *lda);
    r14 = *(A + threadIdx.x + 14 *lda);
    r15 = *(A + threadIdx.x + 15 *lda);
 
    for (int i = 0; i < T; i++) {
      //i+j
      //a = *(A + (i + j) * lda);
      //b1 = cache[i * 2]        
      //b2 = cache[i * 2 + 1]     
      
      temp1 += cacheA[threadIdx.x +i * T] * cacheB[i];
     //temp2 += cacheA[threadIdx.x +i * T] * cacheB[i * 2 + 1];
    }

    cacheA[0] = r0;
    cacheA[1] = r1;
    cacheA[2] = r2;
    cacheA[3] = r3;
    cacheA[4] = r4;
    cacheA[5] = r5;
    cacheA[6] = r6;
    cacheA[7] = r7;

    cacheA[8] = r8;
    cacheA[9] = r9;
    cacheA[10] = r10;
    cacheA[11] = r11;
    cacheA[12] = r12;
    cacheA[13] = r13;
    cacheA[14] = r14;
    cacheA[15] = r15;

  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;

}




__global__ void
dgemm_kernel4_1(int m, int n, int k, int T, double * A, int lda, double * B, int ldb, double * C, int ldc)
{
  // store B (T * 2)                                                                                                                                                                                                                                                                       
  extern __shared__ double cache[];
 
  double * cacheA = cache;
  double * cacheB = cache + T * (T / 4); //32 threads * 8 elements

  //determine the row to process                                                                                                                                                                                                                                                           
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  A = A + idx;
  double temp1 = 0;
  //double temp2 = 0;
  double a = 0;
  //double b1 = 0；                                                                                                                                                                                                                                                                        
  //double b2 = 0;                                                                                                                                                                                                                                                                         

  //prefectch A 
  int t = T / 4;
  for (int i = 0; i < t; i++){
    cacheA[threadIdx.x + i * T] = *(A + i * lda);
  }
  //printf("%d: %f, %f\n", threadIdx.x, cacheA[threadIdx.x + 0 * T], cacheA[threadIdx.x + 1 * T]);
  __syncthreads();

  double r0, r1, r2,r3;//,r4,r5,r6,r7;
  double * orgB = B;
  double * orgA = A;
  for (int j = 0; j < k; j += T){ 
    B = orgB + j;
    __syncthreads();
    cacheB[threadIdx.x] = *(B + threadIdx.x);
    //cacheB[threadIdx.x * 2 + 1] = *(B + threadIdx.x + ldb);
    __syncthreads();
    //printf("[iter=%d]%d: %f, %f\n", j, threadIdx.x, cacheB[threadIdx.x * 2], cacheB[threadIdx.x * 2 + 1]); 

    A = orgA + j * lda;
    for (int l = 0; l < 4; l++){
      
      r0 = *(A + (0 + (l+1) * t) *lda);
      r1 = *(A + (1 + (l+1) * t) *lda);
      r2 = *(A + (2 + (l+1) * t) *lda);
      r3 = *(A + (3 + (l+1) * t) *lda);
      /*r4 = *(A + (4 + (l+1) * t) *lda);
      r5 = *(A + (5 + (l+1) * t) *lda);
      r6 = *(A + (6 + (l+1) * t) *lda);
      r7 = *(A + (7 + (l+1) * t) *lda);
      */
      //__syncthreads();
      //printf("[iter=%d]%d: %f, %f\n", j+l*t, threadIdx.x, cacheA[threadIdx.x + 0 * T], cacheA[threadIdx.x + 1 * T]);
      for (int i = 0; i < t; i++) {
	       temp1 += cacheA[threadIdx.x +i * T] * cacheB[t * l + i ];
	       //temp2 += cacheA[threadIdx.x +i * T] * cacheB[t * l + i * 2 + 1];
      }
      
      cacheA[threadIdx.x + 0 * T] = r0;
      cacheA[threadIdx.x + 1 * T] = r1;
      cacheA[threadIdx.x + 2 * T] = r2;
      cacheA[threadIdx.x + 3 * T] = r3;
      /*cacheA[threadIdx.x + 4 * T] = r4;
      cacheA[threadIdx.x + 5 * T] = r5;
      cacheA[threadIdx.x + 6 * T] = r6;
      cacheA[threadIdx.x + 7 * T] = r7;
      */
    }
  }
  *(C + 0 * ldc + idx) = temp1;
  //*(C + 1 * ldc + idx) = temp2;
    
}
